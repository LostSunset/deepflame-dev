#include "dfCSRPreconditioner.H"
#include "dfSolverOpBase.H"

// kernel functions for PBiCGStab solver

void GAMGCSRPreconditioner::initialize
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata, int agglomeration_level
)
{
    std::cout << "*** call in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    // for(int leveli=0; leveli<agglomeration_level; leveli++)
    // {
    //     std::cout << "malloc leveli: " << leveli << std::endl;
    //     // iteration data
    //     checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_CorrFields, GAMGdata[leveli].nCell * sizeof(double)));
    //     checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_Sources,    GAMGdata[leveli].nCell * sizeof(double)));

    //     checkCudaErrors(hipMemset(GAMGdata[leveli].d_CorrFields, 0, GAMGdata[leveli].nCell * sizeof(double)));
    //     checkCudaErrors(hipMemset(GAMGdata[leveli].d_Sources,    0, GAMGdata[leveli].nCell * sizeof(double)));

    //     // temp data for reduce
    //     // checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorNum,   GAMGdata[leveli].nCell * sizeof(double)));
    //     // checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorDenom, GAMGdata[leveli].nCell * sizeof(double)));
    // }
};

void GAMGCSRPreconditioner::agglomerateMatrix
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::agglomerateMatrix " << std::endl;
    int leveli=0;
    do
    {
        std::cout << "  level: " << leveli << ", in cell: " << GAMGdata_[leveli].nCell
                                           << ", out cell: " << GAMGdata_[leveli+1].nCell << std::endl;

        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_diag, GAMGdata_[leveli+1].d_diag);

        restrictMatrixGPU(dataBase.stream, GAMGdata_[leveli].nFace, 
                        GAMGdata_[leveli].d_faceRestrictMap, GAMGdata_[leveli].d_faceFlipMap,
                        GAMGdata_[leveli].d_upper, GAMGdata_[leveli].d_lower,
                        GAMGdata_[leveli+1].d_upper, GAMGdata_[leveli+1].d_lower, GAMGdata_[leveli+1].d_diag);

        //TODO: calculate interface & interfaceCoef on coarseGrid...

        leveli++; // goto coarser level

    } while( leveli < agglomeration_level-1 );
};

void GAMGCSRPreconditioner::fine2coarse
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::fine2coarse " << std::endl;
    for(int leveli=startLevel; leveli<=endLevel; leveli++)
    {

    }

};

void GAMGCSRPreconditioner::coarse2fine
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::coarse2fine " << std::endl;
    for(int leveli=startLevel; leveli>=endLevel; leveli--)
    {

    }

};

void GAMGCSRPreconditioner::Vcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::Vcycle " << std::endl;
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1);
    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0);
};

void GAMGCSRPreconditioner::precondition
(
    double *wA,
    const double *rA
)
{
    // Implement the GAMG precondition procedure here
    std::cout << "********* call in GAMGCSRPreconditioner::precondition " << std::endl;
};