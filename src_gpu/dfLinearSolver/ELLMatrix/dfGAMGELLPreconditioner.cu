#include "dfELLPreconditioner.H"
#include "dfSolverOpBase.H"

#define nSweeps 2

// kernel functions for PCG solver

void GAMGELLPreconditioner::initCycle
(
    GAMGStruct *GAMGdata, int agglomeration_level                                                                
)
{
    std::cout << "*** call in GAMGELLPreconditioner::initCycle " << std::endl;
    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {                                 
        checkCudaErrors(hipMemset(GAMGdata[leveli].d_CorrFields, 0, GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMemset(GAMGdata[leveli].d_Sources, 0, GAMGdata[leveli].nCell*sizeof(double)));
    }
    std::cout << "*** end in GAMGELLPreconditioner::initCycle " << std::endl;
    std::cout << "*********************************************************** " << std::endl;
};

void GAMGELLPreconditioner::initialize
(
    GAMGStruct *GAMGdata, int agglomeration_level
)
{
    std::cout << "*** call in GAMGELLPreconditioner::initialize(): init Vcycle " << std::endl;

    // Jacobi Smoother
    smoother = new ELLJacobiSmoother();

    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {
        std::cout << "   malloc leveli: " << leveli << std::endl;
        // matrix data                                      
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_lower, GAMGdata[leveli].nFace * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_upper, GAMGdata[leveli].nFace * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_diag,  GAMGdata[leveli].nCell * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_lowerAddr, GAMGdata[leveli].nFace * sizeof(int)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_upperAddr, GAMGdata[leveli].nFace * sizeof(int)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_ell_cols, GAMGdata[leveli].nCell * GAMGdata[leveli].ell_row_maxcount * sizeof(int)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_ell_values, GAMGdata[leveli].nCell * GAMGdata[leveli].ell_row_maxcount * sizeof(double)));

        // iteration data
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_CorrFields, GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_Sources,    GAMGdata[leveli].nCell*sizeof(double)));

        // temp data for reduce
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_AcfField,           GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_preSmoothField,     GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorNum,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorDenom, GAMGdata[leveli].nCell*sizeof(double)));
    }
    std::cout << "*** end in GAMGELLPreconditioner::initialize(): init Vcycle " << std::endl;
    std::cout << "*********************************************************** " << std::endl;
};

void GAMGELLPreconditioner::freeInitialize
(
    GAMGStruct *GAMGdata, int agglomeration_level
)
{
    std::cout << "*** call in GAMGELLPreconditioner::initialize(): init Vcycle " << std::endl;
    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {
        std::cout << "   malloc leveli: " << leveli << std::endl;
        // matrix data                                      
        checkCudaErrors(hipFree(GAMGdata[leveli].d_lower));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_upper));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_diag));       

        // iteration data
        checkCudaErrors(hipFree(GAMGdata[leveli].d_CorrFields));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_Sources));

        // temp data for reduce
        checkCudaErrors(hipFree(GAMGdata[leveli].d_AcfField));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_preSmoothField));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_scalingFactorNum));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_scalingFactorDenom));
    }
    std::cout << "*** end in GAMGELLPreconditioner::initialize(): init Vcycle " << std::endl;
    std::cout << "*********************************************************** " << std::endl;
};

void GAMGELLPreconditioner::agglomerateMatrix
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    std::cout << "********* call in GAMGELLPreconditioner::agglomerateMatrix " << std::endl;
    for(int leveli=0; leveli<agglomeration_level-1; leveli++)
    {
        std::cout << "  level: " << leveli << ", in cell: " << GAMGdata_[leveli].nCell
                                           << ", out cell: " << GAMGdata_[leveli+1].nCell << std::endl;

        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_diag, GAMGdata_[leveli+1].d_diag);

        restrictMatrixGPU(dataBase.stream, GAMGdata_[leveli].nFace, 
                        GAMGdata_[leveli].d_faceRestrictMap, GAMGdata_[leveli].d_faceFlipMap,
                        GAMGdata_[leveli].d_upper, GAMGdata_[leveli].d_lower,
                        GAMGdata_[leveli+1].d_upper, GAMGdata_[leveli+1].d_lower, GAMGdata_[leveli+1].d_diag);

#ifdef PARALLEL_
        // agglomerateInterfaceCoefficients
        for(int patchi=0; patchi<GAMGdata_[leveli].nPatchFaces.size(); patchi++)
        {
            if (GAMGdata_[leveli].nPatchFaces[patchi] > 0)
            {
                restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nPatchFaces[patchi], 
                                GAMGdata_[leveli].d_patchFaceRestrictMap[patchi], 
                                GAMGdata_[leveli].d_interfaceBouCoeffs[patchi], 
                                GAMGdata_[leveli+1].d_interfaceBouCoeffs[patchi]);

                restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nPatchFaces[patchi], 
                                GAMGdata_[leveli].d_patchFaceRestrictMap[patchi], 
                                GAMGdata_[leveli].d_interfaceIntCoeffs[patchi], 
                                GAMGdata_[leveli+1].d_interfaceIntCoeffs[patchi]);
            }
        }
#endif
    }
};

void GAMGELLPreconditioner::fine2coarse
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    bool scaleCorrection = true;

    std::cout << "   ****** call in GAMGELLPreconditioner::fine2coarse " << std::endl;
    for(int leveli=startLevel; leveli<endLevel; leveli++)
    {
        std::cout << "  this level: " << leveli << ", restrict source for coarser level " << std::endl;

        //Purpose: get next level (leveli+1) source
        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_Sources, GAMGdata_[leveli+1].d_Sources);

        //Purpose: coarseCorrFields[leveli] = 0.0;
        checkCudaErrors(hipMemset(GAMGdata_[leveli+1].d_CorrFields, 0, GAMGdata_[leveli+1].nCell*sizeof(double)));

        //Purpose: Smooth [ A * Corr = Source ] to get d_CorrFields for leveli+1
        //TODO: write nSweeps 
        smoother->smooth(dataBase.stream, nSweeps, GAMGdata_[leveli+1].nCell, GAMGdata_[leveli+1].d_CorrFields, 
                            GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].ell_row_maxcount, GAMGdata_[leveli+1].d_ell_cols,
                            GAMGdata_[leveli+1].d_ell_values, GAMGdata_[leveli+1].d_diag);

        if (leveli < endLevel - 1)
        {
            //Purpose: scale d_CorrFields leveli+1, if (matrix.symmetric())
            if (scaleCorrection) 
            {
                scaleFieldGPU_ell( dataBase, GAMGdata_[leveli+1].nCell, 
                    GAMGdata_[leveli+1].d_CorrFields, GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].d_AcfField, 
                    GAMGdata_[leveli+1].d_diag, GAMGdata_[leveli+1].ell_row_maxcount,
                    GAMGdata_[leveli+1].d_ell_cols, GAMGdata_[leveli+1].d_ell_values, 
                    GAMGdata_[leveli+1].d_interfaceIntCoeffs, GAMGdata_[leveli+1].d_interfaceBouCoeffs,
                    GAMGdata_[leveli+1].d_faceCells, GAMGdata_[leveli+1].nPatchFaces, 
                    GAMGdata_[leveli+1].d_scalingFactorNum, GAMGdata_[leveli+1].d_scalingFactorDenom );
            }

            //Purpose: get Acf = A * Corr & GAMGdata_[leveli+1].d_Sources -= Acf
            updateSourceFieldGPU_ell( dataBase, GAMGdata_[leveli+1].nCell, 
                                GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].d_AcfField, GAMGdata_[leveli+1].d_CorrFields,
                                GAMGdata_[leveli+1].d_diag, GAMGdata_[leveli+1].ell_row_maxcount,
                                GAMGdata_[leveli+1].d_ell_cols, GAMGdata_[leveli+1].d_ell_values, 
                                GAMGdata_[leveli+1].d_interfaceIntCoeffs, GAMGdata_[leveli+1].d_interfaceBouCoeffs,
                                GAMGdata_[leveli+1].d_faceCells, GAMGdata_[leveli+1].nPatchFaces);
        }    
    }
};

void GAMGELLPreconditioner::coarse2fine
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    bool interpolateCorrection = false;
    bool scaleCorrection = true;

    std::cout << "   ****** call in GAMGELLPreconditioner::coarse2fine " << std::endl;
    for(int leveli=startLevel; leveli>endLevel; leveli--)
    {
        std::cout << "  this level: " << leveli << ", prolong correct for finer level " << std::endl;

        //Purpose: preSmoothedCoarseCorrField = MGCorrFields[leveli-1];
        checkCudaErrors(hipMemcpyAsync(GAMGdata_[leveli-1].d_preSmoothField, GAMGdata_[leveli-1].d_CorrFields, 
                                        GAMGdata_[leveli-1].nCell*sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream));

        //Purpose: get next level (leveli-1) corr
        prolongFieldGPU(dataBase.stream, GAMGdata_[leveli-1].nCell, 
                        GAMGdata_[leveli-1].d_restrictMap, 
                        GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli].d_CorrFields);

        if (interpolateCorrection)
        {
            //Purpose: interpolate correctionField for next level (leveli-1)
            interpolateFieldGPU_ell(dataBase, GAMGdata_[leveli-1].nCell, GAMGdata_[leveli].nCell, 
                    GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_AcfField, 
                    GAMGdata_[leveli-1].d_diag, GAMGdata_[leveli-1].ell_row_maxcount,
                    GAMGdata_[leveli-1].d_ell_cols, GAMGdata_[leveli-1].d_ell_values,  
                    GAMGdata_[leveli-1].d_interfaceIntCoeffs, GAMGdata_[leveli-1].d_interfaceBouCoeffs, 
                    GAMGdata_[leveli-1].d_faceCells, GAMGdata_[leveli-1].nPatchFaces,
                    GAMGdata_[leveli-1].d_restrictMap, GAMGdata_[leveli].d_CorrFields);
        }

        if (leveli < startLevel && scaleCorrection)
        {
            //Purpose: scale d_CorrFields leveli-1, if (matrix.symmetric())
            scaleFieldGPU_ell( dataBase, GAMGdata_[leveli-1].nCell, 
                GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_Sources, GAMGdata_[leveli-1].d_AcfField, 
                GAMGdata_[leveli-1].d_diag, GAMGdata_[leveli-1].ell_row_maxcount,
                GAMGdata_[leveli-1].d_ell_cols, GAMGdata_[leveli-1].d_ell_values,
                GAMGdata_[leveli-1].d_interfaceIntCoeffs, GAMGdata_[leveli-1].d_interfaceBouCoeffs,
                GAMGdata_[leveli-1].d_faceCells, GAMGdata_[leveli-1].nPatchFaces, 
                GAMGdata_[leveli-1].d_scalingFactorNum, GAMGdata_[leveli-1].d_scalingFactorDenom );
        }
        
        if (leveli > endLevel + 1)
        {
            //Purpose: MGCorrFields[leveli] += preSmoothedCoarseCorrField;
            updateCorrFieldGPU( dataBase.stream, GAMGdata_[leveli-1].nCell, 
                                GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_preSmoothField);

            //Purpose: Smooth [ A * Corr = Source ] to get d_CorrFields for leveli-1
            //TODO: write nSweeps
            smoother->smooth(dataBase.stream, nSweeps, GAMGdata_[leveli-1].nCell, GAMGdata_[leveli-1].d_CorrFields, 
                    GAMGdata_[leveli-1].d_Sources, GAMGdata_[leveli-1].ell_row_maxcount, GAMGdata_[leveli-1].d_ell_cols,
                    GAMGdata_[leveli-1].d_ell_values, GAMGdata_[leveli-1].d_diag);

        }
    }
};

void GAMGELLPreconditioner::directSolveCoarsest
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    bool solveCoarsest = false;
    if (solveCoarsest)
    {
        std::cout << "   ****** call in GAMGELLPreconditioner::directSolveCoarsest " << std::endl;
        if (GAMGdata_[agglomeration_level-1].nCell == 1)
        {
            //directSolve1x1
            directSolve1x1GPU(dataBase.stream, 
                                GAMGdata_[agglomeration_level-1].d_diag, 
                                GAMGdata_[agglomeration_level-1].d_CorrFields, 
                                GAMGdata_[agglomeration_level-1].d_Sources);
        }
        else if (GAMGdata_[agglomeration_level-1].nCell == 4)
        {
            //directSolve4x4
            directSolve4x4GPU(dataBase.stream, 
                        GAMGdata_[agglomeration_level-1].d_diag, 
                        GAMGdata_[agglomeration_level-1].d_upper, 
                        GAMGdata_[agglomeration_level-1].d_lower, 
                        GAMGdata_[agglomeration_level-1].d_CorrFields, 
                        GAMGdata_[agglomeration_level-1].d_Sources);
        }
        else
        {
            std::cout << "*** Unsupported dimension for aggregation amg level ..."<< std::endl;
        }
    }
};

void GAMGELLPreconditioner::Vcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0);
};

void GAMGELLPreconditioner::Wcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, agglomeration_level-2);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-2, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 1);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 1, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, agglomeration_level-2);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-2, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0);
};

void GAMGELLPreconditioner::Fcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, agglomeration_level-2);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-2, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 1);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 1, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0);
};

void GAMGELLPreconditioner::precondition
(
    double *psi,
    const double *finestResidual,
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{

    std::cout << "******************************************************" << std::endl;
    std::cout << "********* call in GAMGELLPreconditioner::precondition " << std::endl;

    //TODO: get nVcycles from control files
    int nVcycles_ = 1; 
    initCycle(GAMGdata_, agglomeration_level);

    // Purpose: wA = 0.0;
    checkCudaErrors(hipMemset(psi, 0, GAMGdata_[0].nCell*sizeof(double)));

    // Purpose: set GAMGdata_[0].d_Sources
    checkCudaErrors(hipMemcpyAsync(GAMGdata_[0].d_Sources, finestResidual, GAMGdata_[0].nCell*sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream));

    for (int cycle=0; cycle<nVcycles_; cycle++)
    {
        // Purpose: do Vcycle calculation
        Vcycle(dataBase, GAMGdata_, agglomeration_level);

        // Purpose: use GAMGdata_[0].d_CorrFields to update psi
        updateCorrFieldGPU( dataBase.stream, GAMGdata_[0].nCell, psi, GAMGdata_[0].d_CorrFields);

        //add smoother for leveli=0, nFinestSweeps_
        //TODO: write nSweeps 
        smoother->smooth(dataBase.stream, nSweeps, GAMGdata_[0].nCell, psi, 
                    GAMGdata_[0].d_Sources, GAMGdata_[0].ell_row_maxcount, GAMGdata_[0].d_ell_cols,
                    GAMGdata_[0].d_ell_values, GAMGdata_[0].d_diag);


        if (cycle < nVcycles_-1)
        {
            // Purpose: Calculate finest level residual field to update finestResidual
            updateSourceFieldGPU_ell( dataBase, GAMGdata_[0].nCell, 
                                GAMGdata_[0].d_Sources, GAMGdata_[0].d_AcfField, psi,
                                GAMGdata_[0].d_diag, GAMGdata_[0].ell_row_maxcount, 
                                GAMGdata_[0].d_ell_cols, GAMGdata_[0].d_ell_values, 
                                GAMGdata_[0].d_interfaceIntCoeffs, GAMGdata_[0].d_interfaceBouCoeffs,
                                GAMGdata_[0].d_faceCells, GAMGdata_[0].nPatchFaces);
        }
    }
    std::cout << "********** end in GAMGELLPreconditioner::precondition " << std::endl;
    std::cout << "******************************************************" << std::endl;
};

