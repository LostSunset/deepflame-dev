#include "dfCSRPreconditioner.H"
#include "dfSolverOpBase.H"

// kernel functions for PBiCGStab solver

void GAMGCSRPreconditioner::initialize
(
    GAMGStruct *GAMGdata, int agglomeration_level
)
{
    std::cout << "*** call in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {
        std::cout << "   malloc leveli: " << leveli << std::endl;
        // matrix data                                      
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_lower, GAMGdata[leveli].nFace * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_upper, GAMGdata[leveli].nFace * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_diag, GAMGdata[leveli].nCell * sizeof(double)));       

        // iteration data
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_CorrFields, GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_Sources,    GAMGdata[leveli].nCell*sizeof(double)));

        // temp data for reduce
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_AcfField,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_preSmoothField,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorNum,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorDenom, GAMGdata[leveli].nCell*sizeof(double)));
    }
    std::cout << "*** end in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    std::cout << "*********************************************************** " << std::endl;
};

void GAMGCSRPreconditioner::agglomerateMatrix
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::agglomerateMatrix " << std::endl;
    for(int leveli=0; leveli<agglomeration_level-1; leveli++)
    {
        std::cout << "  level: " << leveli << ", in cell: " << GAMGdata_[leveli].nCell
                                           << ", out cell: " << GAMGdata_[leveli+1].nCell << std::endl;

        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_diag, GAMGdata_[leveli+1].d_diag);

        restrictMatrixGPU(dataBase.stream, GAMGdata_[leveli].nFace, 
                        GAMGdata_[leveli].d_faceRestrictMap, GAMGdata_[leveli].d_faceFlipMap,
                        GAMGdata_[leveli].d_upper, GAMGdata_[leveli].d_lower,
                        GAMGdata_[leveli+1].d_upper, GAMGdata_[leveli+1].d_lower, GAMGdata_[leveli+1].d_diag);

#ifndef PARALLEL_
        // agglomerateInterfaceCoefficients
        for(int patchi=0; patchi<GAMGdata_[leveli].nPatchFaces.size(); patchi++)
        {
            if (GAMGdata_[leveli].nPatchFaces[patchi] > 0)
            {
                restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nPatchFaces[patchi], 
                                GAMGdata_[leveli].d_patchFaceRestrictMap[patchi], 
                                GAMGdata_[leveli].d_interfaceBouCoeffs[patchi], 
                                GAMGdata_[leveli+1].d_interfaceBouCoeffs[patchi]);

                restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nPatchFaces[patchi], 
                                GAMGdata_[leveli].d_patchFaceRestrictMap[patchi], 
                                GAMGdata_[leveli].d_interfaceIntCoeffs[patchi], 
                                GAMGdata_[leveli+1].d_interfaceIntCoeffs[patchi]);
            }
        }
#endif
    }
};

void GAMGCSRPreconditioner::fine2coarse
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    std::cout << "   ****** call in GAMGCSRPreconditioner::fine2coarse " << std::endl;
    for(int leveli=startLevel; leveli<endLevel; leveli++)
    {
        std::cout << "  this level: " << leveli << ", restrict source for coarser level " << std::endl;

        //Purpose: get next level (leveli+1) source
        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_Sources, GAMGdata_[leveli+1].d_Sources);
        
        //Purpose: coarseCorrFields[leveli] = 0.0;
        checkCudaErrors(hipMemset(GAMGdata_[leveli+1].d_CorrFields, 0, GAMGdata_[leveli+1].nCell*sizeof(double)));

        //Purpose: Smooth [ A * Corr = Source ] to get d_CorrFields for leveli+1
        //TODO: add smoother here

        if (leveli < endLevel - 1)
        {
            //Purpose: scale d_CorrFields leveli+1, if (matrix.symmetric())
            //TODO: add scale here, (need calc Acf in scale) 

            //Purpose: spmv to get Acf = A * Corr
            //TODO: add Amul to get Acf

            //Purpose: GAMGdata_[leveli+1].d_Sources -= Acf
            updateSourceFieldGPU( dataBase.stream, GAMGdata_[leveli+1].nCell, 
                                  GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].d_AcfField);
        }    
    }
};

void GAMGCSRPreconditioner::coarse2fine
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    std::cout << "   ****** call in GAMGCSRPreconditioner::coarse2fine " << std::endl;
    for(int leveli=startLevel; leveli>endLevel; leveli--)
    {
        std::cout << "  this level: " << leveli << ", prolong correct for finer level " << std::endl;

        //Purpose: preSmoothedCoarseCorrField = MGCorrFields[leveli-1];
        checkCudaErrors(hipMemcpyAsync(GAMGdata_[leveli-1].d_preSmoothField, GAMGdata_[leveli-1].d_CorrFields, 
                                        GAMGdata_[leveli-1].nCell*sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream));

        //Purpose: get next level (leveli-1) corr
        prolongFieldGPU(dataBase.stream, GAMGdata_[leveli-1].nCell, 
                        GAMGdata_[leveli-1].d_restrictMap, 
                        GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli].d_CorrFields);

        if (leveli < startLevel - 1)
        {
            //Purpose: scale d_CorrFields leveli-1, if (matrix.symmetric())
            //TODO: add scale here
        }
        
        if (leveli > endLevel + 1)
        {
            //Purpose: MGCorrFields[leveli] += preSmoothedCoarseCorrField;
            updateCorrFieldGPU( dataBase.stream, GAMGdata_[leveli-1].nCell, 
                                GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_preSmoothField);

            //Purpose: Smooth [ A * Corr = Source ] to get d_CorrFields for leveli-1
            //TODO: add smoother here for leveli-1
        }
    }
};

void GAMGCSRPreconditioner::directSolveCoarsest
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    bool solveCoarsest = false;
    if (solveCoarsest)
    {
        std::cout << "   ****** call in GAMGCSRPreconditioner::directSolveCoarsest " << std::endl;
        if (GAMGdata_[agglomeration_level-1].nCell == 1)
        {
            //directSolve1x1
            directSolve1x1GPU(dataBase.stream, GAMGdata_[agglomeration_level-1].nCell, 
                                GAMGdata_[agglomeration_level-1].d_diag, 
                                GAMGdata_[agglomeration_level-1].d_CorrFields, 
                                GAMGdata_[agglomeration_level-1].d_Sources);
        }
        else if (GAMGdata_[agglomeration_level-1].nCell == 4)
        {
            //directSolve4x4
        }
        else
        {
            std::cout << "*** Unsupported dimension for aggregation amg level ..."<< std::endl;
        }
    }
};

void GAMGCSRPreconditioner::Vcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0);
};

void GAMGCSRPreconditioner::precondition
(
    double *psi,
    const double *finestResidual,
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    std::cout << "******************************************************" << std::endl;
    std::cout << "********* call in GAMGCSRPreconditioner::precondition " << std::endl;

    // wA = 0.0;
    checkCudaErrors(hipMemset(psi, 0, GAMGdata_[0].nCell*sizeof(double)));

    //TODO: get nVcycles from control files
    int nVcycles_ = 1; 

    // set GAMGdata_[0].d_Sources
    checkCudaErrors(hipMemcpyAsync(GAMGdata_[0].d_Sources, finestResidual, GAMGdata_[0].nCell*sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream));

    for (int cycle=0; cycle<nVcycles_; cycle++)
    {
        // start Vcycle
        Vcycle(dataBase, GAMGdata_, agglomeration_level);

        // use GAMGdata_[0].d_CorrFields to update psi
        updateCorrFieldGPU( dataBase.stream, GAMGdata_[0].nCell, psi, GAMGdata_[0].d_CorrFields);

        //TODO: add smoother for leveli=0, nFinestSweeps_

        if (cycle < nVcycles_-1)
        {
            // TODO: Calculate finest level residual field to update finestResidual
            // matrix_.Amul(AwA, wA, interfaceBouCoeffs_, interfaces_, cmpt);

            updateSourceFieldGPU(dataBase.stream, GAMGdata_[0].nCell, GAMGdata_[0].d_Sources, GAMGdata_[0].d_AcfField);
        }
    }
    std::cout << "********** end in GAMGCSRPreconditioner::precondition " << std::endl;
    std::cout << "******************************************************" << std::endl;
};