#include "hip/hip_runtime.h"
#include "dfELLSmoother.H"
#include "dfSolverOpBase.H"

#define PARALLEL_

__global__ void ellJacobiSmooth
(
    int nCells,
    double* psi,
    double* psiCopyPtr,
    double* source,
    int ell_row_maxcount,
    int* d_ell_cols, 
    double* d_ell_values,
    double* diagPtr
)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= nCells)
        return;
    
    double sum = source[index];
    int offset = ell_row_maxcount * index;
    for(int r = 0; r < ell_row_maxcount; r++){
        sum -= d_ell_values[r + offset] * psiCopyPtr[d_ell_cols[r + offset]];
    }
    psi[index] = sum / diagPtr[index];
}

void ELLJacobiSmoother::smooth
(
    hipStream_t stream,
    int nSweeps,
    int nCells,
    double* psi,
    double* source,
    int ell_row_maxcount,
    int* d_ell_cols, 
    double* d_ell_values,
    double* diagPtr,
    // PARALLEL_
    const dfMatrixDataBase& dataBase,
    double* scalarSendBufList_, 
    double* scalarRecvBufList_,
    double** interfaceBouCoeffs,
    int** faceCells, std::vector<int> nPatchFaces
)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (nCells + threads_per_block - 1) / threads_per_block;
    
    double* bPrime;
    hipMalloc(&bPrime, nCells * sizeof(double));

    for (int sweep=0; sweep<nSweeps; sweep++)
    {
        hipMemcpyAsync(bPrime, source, nCells * sizeof(double), hipMemcpyDeviceToDevice, stream);

#ifdef PARALLEL_   
        // sign = -1 for negate()
        // --- initMatrixInterfaces & updateMatrixInterfaces ---
        updateMatrixInterfaceCoeffs(
            dataBase.stream, dataBase.neighbProcNo, dataBase.nccl_comm,
            nPatchFaces, psi, bPrime, 
            scalarSendBufList_, scalarRecvBufList_,
            interfaceBouCoeffs, faceCells, -1.0);
#endif

        double* psiCopyPtr;
        hipMallocAsync(&psiCopyPtr, nCells * sizeof(double), stream);
        hipMemcpyAsync(psiCopyPtr, psi, nCells * sizeof(double), hipMemcpyDeviceToDevice, stream);
    
        ellJacobiSmooth<<<blocks_per_grid, threads_per_block, 0, stream>>>
            (nCells, psi, psiCopyPtr, bPrime, ell_row_maxcount, d_ell_cols, d_ell_values, diagPtr);
        checkCudaErrors(hipStreamSynchronize(stream));

    }
};