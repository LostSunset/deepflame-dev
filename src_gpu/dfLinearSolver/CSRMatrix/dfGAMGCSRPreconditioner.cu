#include "dfCSRPreconditioner.H"
#include "dfSolverOpBase.H"
#include <nvtx3/nvToolsExt.h>

#define PARALLEL_

#define nSweeps 2

// kernel functions for PCG solver

void GAMGCSRPreconditioner::initCycle
(
    GAMGStruct *GAMGdata, int agglomeration_level                                                                
)
{
    // std::cout << "*** call in GAMGCSRPreconditioner::initCycle " << std::endl;
    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {                                 
        checkCudaErrors(hipMemset(GAMGdata[leveli].d_CorrFields, 0, GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMemset(GAMGdata[leveli].d_Sources, 0, GAMGdata[leveli].nCell*sizeof(double)));
    }
    // std::cout << "*** end in GAMGCSRPreconditioner::initCycle " << std::endl;
    // std::cout << "*********************************************************** " << std::endl;
};

void GAMGCSRPreconditioner::initialize
(
    const dfMatrixDataBase &dataBase, GAMGStruct *GAMGdata, int agglomeration_level
)
{
    std::cout << "*** call in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;

    // Jacobi Smoother
    smoother = new CSRJacobiSmoother();

    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {
        std::cout << "   malloc leveli: " << leveli << std::endl;
        // matrix data                                      
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_lower, GAMGdata[leveli].nFace * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_upper, GAMGdata[leveli].nFace * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_diag,  GAMGdata[leveli].nCell * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_lowerAddr, GAMGdata[leveli].nFace * sizeof(int)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_upperAddr, GAMGdata[leveli].nFace * sizeof(int)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_off_diag_value, GAMGdata[leveli].nFace * 2 * sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_csr_row_index_no_diag, (GAMGdata[leveli].nCell + 1) * sizeof(int)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_csr_col_index_no_diag, GAMGdata[leveli].nFace * 2 * sizeof(int)));
        checkCudaErrors(hipMemcpy(GAMGdata[leveli].d_csr_row_index_no_diag, dataBase.h_csr_row_index_no_diag[leveli], (GAMGdata[leveli].nCell + 1) * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(GAMGdata[leveli].d_csr_col_index_no_diag, dataBase.h_csr_col_index_no_diag[leveli], GAMGdata[leveli].nFace * 2 * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(GAMGdata[leveli].d_lowerAddr, &GAMGdata[leveli].lowerAddr[0], GAMGdata[leveli].nFace * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(GAMGdata[leveli].d_upperAddr, &GAMGdata[leveli].upperAddr[0], GAMGdata[leveli].nFace * sizeof(int), hipMemcpyHostToDevice));

        // iteration data
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_CorrFields, GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_Sources,    GAMGdata[leveli].nCell*sizeof(double)));

        // temp data for reduce
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_AcfField,           GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_preSmoothField,     GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorNum,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorDenom, GAMGdata[leveli].nCell*sizeof(double)));
    }
    std::cout << "*** end in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    std::cout << "*********************************************************** " << std::endl;
};

void GAMGCSRPreconditioner::freeInitialize
(
    GAMGStruct *GAMGdata, int agglomeration_level
)
{
    std::cout << "*** call in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {
        std::cout << "   malloc leveli: " << leveli << std::endl;
        // matrix data                                      
        checkCudaErrors(hipFree(GAMGdata[leveli].d_lower));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_upper));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_diag));       

        // iteration data
        checkCudaErrors(hipFree(GAMGdata[leveli].d_CorrFields));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_Sources));

        // temp data for reduce
        checkCudaErrors(hipFree(GAMGdata[leveli].d_AcfField));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_preSmoothField));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_scalingFactorNum));
        checkCudaErrors(hipFree(GAMGdata[leveli].d_scalingFactorDenom));
    }
    std::cout << "*** end in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    std::cout << "*********************************************************** " << std::endl;
};

void GAMGCSRPreconditioner::agglomerateMatrix
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::agglomerateMatrix " << std::endl;
    for(int leveli=0; leveli<agglomeration_level-1; leveli++)
    {
        std::cout << "  level: " << leveli << ", in cell: " << GAMGdata_[leveli].nCell
                                           << ", out cell: " << GAMGdata_[leveli+1].nCell << std::endl;

        nvtxRangePushA("restrictFieldGPU()");
        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_diag, GAMGdata_[leveli+1].d_diag);
        nvtxRangePop();

        nvtxRangePushA("restrictMatrixGPU()");
        restrictMatrixGPU(dataBase.stream, GAMGdata_[leveli].nFace, 
                        GAMGdata_[leveli].d_faceRestrictMap, GAMGdata_[leveli].d_faceFlipMap,
                        GAMGdata_[leveli].d_upper, GAMGdata_[leveli].d_lower,
                        GAMGdata_[leveli+1].d_upper, GAMGdata_[leveli+1].d_lower, GAMGdata_[leveli+1].d_diag);
        nvtxRangePop();

#ifdef PARALLEL_
        // agglomerateInterfaceCoefficients
        for(int patchi=0; patchi<GAMGdata_[leveli].nPatchFaces.size(); patchi++)
        {
            if (GAMGdata_[leveli].nPatchFaces[patchi] > 0)
            {
                restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nPatchFaces[patchi], 
                                GAMGdata_[leveli].d_patchFaceRestrictMap[patchi], 
                                GAMGdata_[leveli].d_interfaceBouCoeffs[patchi], 
                                GAMGdata_[leveli+1].d_interfaceBouCoeffs[patchi]);

                restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nPatchFaces[patchi], 
                                GAMGdata_[leveli].d_patchFaceRestrictMap[patchi], 
                                GAMGdata_[leveli].d_interfaceIntCoeffs[patchi], 
                                GAMGdata_[leveli+1].d_interfaceIntCoeffs[patchi]);
            }
        }
#endif
    }
};

void GAMGCSRPreconditioner::fine2coarse
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel,
    double *scalarSendBufList_, double *scalarRecvBufList_
)
{
    bool scaleCorrection = true;

    std::cout << "   ****** call in GAMGCSRPreconditioner::fine2coarse " << std::endl;
    for(int leveli=startLevel; leveli<endLevel; leveli++)
    {
        std::cout << "  this level: " << leveli << ", restrict source for coarser level " << std::endl;

        //Purpose: get next level (leveli+1) source
        nvtxRangePushA("fine2coarse::restrictFieldGPU()");
        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_Sources, GAMGdata_[leveli+1].d_Sources);
        nvtxRangePop();

        //Purpose: coarseCorrFields[leveli] = 0.0;
        checkCudaErrors(hipMemset(GAMGdata_[leveli+1].d_CorrFields, 0, GAMGdata_[leveli+1].nCell*sizeof(double)));

        //Purpose: Smooth [ A * Corr = Source ] to get d_CorrFields for leveli+1
        //TODO: write nSweeps 
        nvtxRangePushA("fine2coarse::smooth()");
        smoother->smooth(dataBase.stream, nSweeps, GAMGdata_[leveli+1].nCell, GAMGdata_[leveli+1].d_CorrFields, 
                            GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].d_off_diag_value, GAMGdata_[leveli+1].d_csr_row_index_no_diag,
                            GAMGdata_[leveli+1].d_csr_col_index_no_diag, GAMGdata_[leveli+1].d_diag, 
                            dataBase, scalarSendBufList_, scalarRecvBufList_, 
                            GAMGdata_[leveli+1].d_interfaceBouCoeffs, GAMGdata_[leveli+1].d_faceCells, 
                            GAMGdata_[leveli+1].nPatchFaces);
        nvtxRangePop();

        if (leveli < endLevel - 1)
        {
            //Purpose: scale d_CorrFields leveli+1, if (matrix.symmetric())
            if (scaleCorrection) 
            {
                nvtxRangePushA("fine2coarse::scaleFieldGPU()");
                scaleFieldGPU( dataBase, GAMGdata_[leveli+1].nCell, 
                    GAMGdata_[leveli+1].d_CorrFields, GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].d_AcfField, 
                    GAMGdata_[leveli+1].d_diag, GAMGdata_[leveli+1].d_off_diag_value,
                    GAMGdata_[leveli+1].d_csr_row_index_no_diag, GAMGdata_[leveli+1].d_csr_col_index_no_diag, 
                    GAMGdata_[leveli+1].d_interfaceIntCoeffs, GAMGdata_[leveli+1].d_interfaceBouCoeffs,
                    GAMGdata_[leveli+1].d_faceCells, GAMGdata_[leveli+1].nPatchFaces, 
                    GAMGdata_[leveli+1].d_scalingFactorNum, GAMGdata_[leveli+1].d_scalingFactorDenom,
                    scalarSendBufList_, scalarRecvBufList_ );
                nvtxRangePop();
            }

            //Purpose: get Acf = A * Corr & GAMGdata_[leveli+1].d_Sources -= Acf
            nvtxRangePushA("fine2coarse::updateSourceFieldGPU()");
            updateSourceFieldGPU( dataBase, GAMGdata_[leveli+1].nCell, 
                                GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].d_AcfField, GAMGdata_[leveli+1].d_CorrFields,
                                GAMGdata_[leveli+1].d_diag, GAMGdata_[leveli+1].d_off_diag_value, 
                                GAMGdata_[leveli+1].d_csr_row_index_no_diag, GAMGdata_[leveli+1].d_csr_col_index_no_diag, 
                                GAMGdata_[leveli+1].d_interfaceIntCoeffs, GAMGdata_[leveli+1].d_interfaceBouCoeffs,
                                GAMGdata_[leveli+1].d_faceCells, GAMGdata_[leveli+1].nPatchFaces,
                                scalarSendBufList_, scalarRecvBufList_);
            nvtxRangePop();
        }    
    }
};

void GAMGCSRPreconditioner::coarse2fine
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel,
    double *scalarSendBufList_, double *scalarRecvBufList_
)
{
    bool interpolateCorrection = false;
    bool scaleCorrection = true;

    std::cout << "   ****** call in GAMGCSRPreconditioner::coarse2fine " << std::endl;
    for(int leveli=startLevel; leveli>endLevel; leveli--)
    {
        std::cout << "  this level: " << leveli << ", prolong correct for finer level " << std::endl;

        //Purpose: preSmoothedCoarseCorrField = MGCorrFields[leveli-1];
        checkCudaErrors(hipMemcpyAsync(GAMGdata_[leveli-1].d_preSmoothField, GAMGdata_[leveli-1].d_CorrFields, 
                                        GAMGdata_[leveli-1].nCell*sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream));

        //Purpose: get next level (leveli-1) corr
        nvtxRangePushA("fine2coarse::prolongFieldGPU()");
        prolongFieldGPU(dataBase.stream, GAMGdata_[leveli-1].nCell, 
                        GAMGdata_[leveli-1].d_restrictMap, 
                        GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli].d_CorrFields);
        nvtxRangePop();
        
        if (interpolateCorrection)
        {
            //Purpose: interpolate correctionField for next level (leveli-1)
            nvtxRangePushA("fine2coarse::interpolateFieldGPU()");
            interpolateFieldGPU(dataBase, GAMGdata_[leveli-1].nCell, GAMGdata_[leveli].nCell, 
                    GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_AcfField, 
                    GAMGdata_[leveli-1].d_diag, GAMGdata_[leveli-1].d_off_diag_value,
                    GAMGdata_[leveli-1].d_csr_row_index_no_diag, GAMGdata_[leveli-1].d_csr_col_index_no_diag,  
                    GAMGdata_[leveli-1].d_interfaceIntCoeffs, GAMGdata_[leveli-1].d_interfaceBouCoeffs, 
                    GAMGdata_[leveli-1].d_faceCells, GAMGdata_[leveli-1].nPatchFaces,
                    GAMGdata_[leveli-1].d_restrictMap, GAMGdata_[leveli].d_CorrFields,
                    scalarSendBufList_, scalarRecvBufList_);
            nvtxRangePop();
        }

        if (leveli < startLevel && scaleCorrection)
        {
            //Purpose: scale d_CorrFields leveli-1, if (matrix.symmetric())
            nvtxRangePushA("fine2coarse::scaleFieldGPU()");
            scaleFieldGPU( dataBase, GAMGdata_[leveli-1].nCell, 
                GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_Sources, GAMGdata_[leveli-1].d_AcfField, 
                GAMGdata_[leveli-1].d_diag, GAMGdata_[leveli-1].d_off_diag_value,
                GAMGdata_[leveli-1].d_csr_row_index_no_diag, GAMGdata_[leveli-1].d_csr_col_index_no_diag, 
                GAMGdata_[leveli-1].d_interfaceIntCoeffs, GAMGdata_[leveli-1].d_interfaceBouCoeffs,
                GAMGdata_[leveli-1].d_faceCells, GAMGdata_[leveli-1].nPatchFaces, 
                GAMGdata_[leveli-1].d_scalingFactorNum, GAMGdata_[leveli-1].d_scalingFactorDenom,
                scalarSendBufList_, scalarRecvBufList_ );
            nvtxRangePop();
        }
        
        if (leveli > endLevel + 1)
        {
            //Purpose: MGCorrFields[leveli] += preSmoothedCoarseCorrField;
            nvtxRangePushA("fine2coarse::updateCorrFieldGPU()");
            updateCorrFieldGPU( dataBase.stream, GAMGdata_[leveli-1].nCell, 
                                GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_preSmoothField);
            nvtxRangePop();

            //Purpose: Smooth [ A * Corr = Source ] to get d_CorrFields for leveli-1
            //TODO: write nSweeps
            nvtxRangePushA("fine2coarse::smooth()");
            smoother->smooth(dataBase.stream, nSweeps, GAMGdata_[leveli-1].nCell, GAMGdata_[leveli-1].d_CorrFields, 
                    GAMGdata_[leveli-1].d_Sources, GAMGdata_[leveli-1].d_off_diag_value, GAMGdata_[leveli-1].d_csr_row_index_no_diag,
                    GAMGdata_[leveli-1].d_csr_col_index_no_diag, GAMGdata_[leveli-1].d_diag, 
                    dataBase, scalarSendBufList_, scalarRecvBufList_, 
                    GAMGdata_[leveli-1].d_interfaceBouCoeffs, GAMGdata_[leveli-1].d_faceCells, 
                    GAMGdata_[leveli-1].nPatchFaces);
            nvtxRangePop();

        }
    }
};

void GAMGCSRPreconditioner::directSolveCoarsest
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    bool solveCoarsest = false;
    if (solveCoarsest)
    {
        std::cout << "   ****** call in GAMGCSRPreconditioner::directSolveCoarsest " << std::endl;
        if (GAMGdata_[agglomeration_level-1].nCell == 1)
        {
            //directSolve1x1
            directSolve1x1GPU(dataBase.stream, 
                                GAMGdata_[agglomeration_level-1].d_diag, 
                                GAMGdata_[agglomeration_level-1].d_CorrFields, 
                                GAMGdata_[agglomeration_level-1].d_Sources);
        }
        else if (GAMGdata_[agglomeration_level-1].nCell == 4)
        {
            //directSolve4x4
            directSolve4x4GPU(dataBase.stream, 
                        GAMGdata_[agglomeration_level-1].d_diag, 
                        GAMGdata_[agglomeration_level-1].d_upper, 
                        GAMGdata_[agglomeration_level-1].d_lower, 
                        GAMGdata_[agglomeration_level-1].d_CorrFields, 
                        GAMGdata_[agglomeration_level-1].d_Sources);
        }
        else
        {
            std::cout << "*** Unsupported dimension for aggregation amg level ..."<< std::endl;
        }
    }
};

void GAMGCSRPreconditioner::Vcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    double *scalarSendBufList_, double *scalarRecvBufList_
)
{
    nvtxRangePushA("Vcycle::fine2coarse()");
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);
    nvtxRangePop();

    nvtxRangePushA("Vcycle::directSolveCoarsest()");
    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);
    nvtxRangePop();

    nvtxRangePushA("Vcycle::coarse2fine()");
    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0, scalarSendBufList_, scalarRecvBufList_);
    nvtxRangePop();
};

void GAMGCSRPreconditioner::Wcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    double *scalarSendBufList_, double *scalarRecvBufList_
)
{
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, agglomeration_level-2, scalarSendBufList_, scalarRecvBufList_);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-2, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 1, scalarSendBufList_, scalarRecvBufList_);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 1, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, agglomeration_level-2, scalarSendBufList_, scalarRecvBufList_);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-2, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0, scalarSendBufList_, scalarRecvBufList_);
};

void GAMGCSRPreconditioner::Fcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    double *scalarSendBufList_, double *scalarRecvBufList_
)
{
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, agglomeration_level-2, scalarSendBufList_, scalarRecvBufList_);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-2, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 1, scalarSendBufList_, scalarRecvBufList_);

    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 1, agglomeration_level-1, scalarSendBufList_, scalarRecvBufList_);

    directSolveCoarsest(dataBase, GAMGdata_, agglomeration_level);

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0, scalarSendBufList_, scalarRecvBufList_);
};

void GAMGCSRPreconditioner::precondition
(
    double *psi,
    const double *finestResidual,
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    double *scalarSendBufList_, double *scalarRecvBufList_
)
{

    std::cout << "******************************************************" << std::endl;
    std::cout << "********* call in GAMGCSRPreconditioner::precondition " << std::endl;

    //TODO: get nVcycles from control files
    int nVcycles_ = 1; 
    nvtxRangePushA("Precondition::initCycle()");
    initCycle(GAMGdata_, agglomeration_level);
    nvtxRangePop();

    // Purpose: wA = 0.0;
    checkCudaErrors(hipMemset(psi, 0, GAMGdata_[0].nCell*sizeof(double)));

    // Purpose: set GAMGdata_[0].d_Sources
    checkCudaErrors(hipMemcpyAsync(GAMGdata_[0].d_Sources, finestResidual, GAMGdata_[0].nCell*sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream));

    for (int cycle=0; cycle<nVcycles_; cycle++)
    {
        // Purpose: do Vcycle calculation
        nvtxRangePushA("Precondition::Vcycle()");
        Vcycle(dataBase, GAMGdata_, agglomeration_level, scalarSendBufList_, scalarRecvBufList_);
        nvtxRangePop();

        // Purpose: use GAMGdata_[0].d_CorrFields to update psi
        nvtxRangePushA("Precondition::updateCorrFieldGPU()");
        updateCorrFieldGPU( dataBase.stream, GAMGdata_[0].nCell, psi, GAMGdata_[0].d_CorrFields);
        nvtxRangePop();

        //add smoother for leveli=0, nFinestSweeps_
        //TODO: write nSweeps 
        nvtxRangePushA("Precondition::smooth()");
        smoother->smooth(dataBase.stream, nSweeps, GAMGdata_[0].nCell, psi, 
                    GAMGdata_[0].d_Sources, GAMGdata_[0].d_off_diag_value, GAMGdata_[0].d_csr_row_index_no_diag,
                    GAMGdata_[0].d_csr_col_index_no_diag, GAMGdata_[0].d_diag, 
                    dataBase, scalarSendBufList_, scalarRecvBufList_, 
                    GAMGdata_[0].d_interfaceBouCoeffs, GAMGdata_[0].d_faceCells, 
                    GAMGdata_[0].nPatchFaces);
        nvtxRangePop();


        if (cycle < nVcycles_-1)
        {
            // Purpose: Calculate finest level residual field to update finestResidual
            nvtxRangePushA("Precondition::updateSourceFieldGPU()");
            updateSourceFieldGPU( dataBase, GAMGdata_[0].nCell, 
                                GAMGdata_[0].d_Sources, GAMGdata_[0].d_AcfField, psi,
                                GAMGdata_[0].d_diag, GAMGdata_[0].d_off_diag_value, 
                                GAMGdata_[0].d_csr_row_index_no_diag, GAMGdata_[0].d_csr_col_index_no_diag, 
                                GAMGdata_[0].d_interfaceIntCoeffs, GAMGdata_[0].d_interfaceBouCoeffs,
                                GAMGdata_[0].d_faceCells, GAMGdata_[0].nPatchFaces,
                                scalarSendBufList_, scalarRecvBufList_);
            nvtxRangePop();
        }
    }
    std::cout << "********** end in GAMGCSRPreconditioner::precondition " << std::endl;
    std::cout << "******************************************************" << std::endl;
};