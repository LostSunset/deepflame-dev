#include "hip/hip_runtime.h"
#include "dfELLSolver.H"
#include "dfSolverOpBase.H"
#include "dfMatrixDataBase.H"

// #define PARALLEL_
#define PRINT_

// kernel functions for PCG solver
void PCGELLSolver::initialize(const int nCells, const size_t boundary_surface_value_bytes)
{
    // cudamalloc variables related to PCGSolver
    hipMalloc(&d_wA, nCells * sizeof(double));
    hipMalloc(&d_rA, nCells * sizeof(double));
    hipMalloc(&d_pA, nCells * sizeof(double));
    hipMalloc(&d_normFactors_tmp, nCells * sizeof(double));
    hipMalloc(&d_wArA_tmp, nCells * sizeof(double));
    hipMalloc(&d_wApA_tmp, nCells * sizeof(double));
    hipMalloc(&reduce_result, sizeof(double));
    // for parallel
    hipMalloc(&scalarSendBufList_, boundary_surface_value_bytes);
    hipMalloc(&scalarRecvBufList_, boundary_surface_value_bytes);

    // preconditioner

}

void PCGELLSolver::initializeStream(const int nCells, const size_t boundary_surface_value_bytes, hipStream_t stream)
{
    // cudamalloc variables related to PCGSolver
    hipMallocAsync(&d_wA, nCells * sizeof(double), stream);
    hipMallocAsync(&d_rA, nCells * sizeof(double), stream);
    hipMallocAsync(&d_pA, nCells * sizeof(double), stream);
    hipMallocAsync(&d_normFactors_tmp, nCells * sizeof(double), stream);
    hipMallocAsync(&d_wArA_tmp, nCells * sizeof(double), stream);
    hipMallocAsync(&d_wApA_tmp, nCells * sizeof(double), stream);
    hipMallocAsync(&reduce_result, sizeof(double), stream);
    // for parallel
    hipMallocAsync(&scalarSendBufList_, boundary_surface_value_bytes, stream);
    hipMallocAsync(&scalarRecvBufList_, boundary_surface_value_bytes, stream);

    // preconditioner

}

void PCGELLSolver::freeInit(){
    hipFree(d_wA);
    hipFree(d_rA);
    hipFree(d_pA);
    hipFree(d_normFactors_tmp);
    hipFree(d_wArA_tmp);
    hipFree(d_wApA_tmp);
    hipFree(reduce_result);
    hipFree(scalarSendBufList_);
    hipFree(scalarRecvBufList_);
}

void PCGELLSolver::freeInitStream(hipStream_t stream)
{
    hipFreeAsync(d_wA, stream);
    hipFreeAsync(d_rA, stream);
    hipFreeAsync(d_pA, stream);
    hipFreeAsync(d_normFactors_tmp, stream);
    hipFreeAsync(d_wArA_tmp, stream);
    hipFreeAsync(d_wApA_tmp, stream);
    hipFreeAsync(reduce_result, stream);
    hipFreeAsync(scalarSendBufList_, stream);
    hipFreeAsync(scalarRecvBufList_, stream);
}

void PCGELLSolver::initializeGAMG(const int nCells, const size_t boundary_surface_value_bytes,
                    GAMGStruct *GAMGdata_, int agglomeration_level)
{
    // cudamalloc variables related to PCGSolver
    hipMalloc(&d_wA, nCells * sizeof(double));
    hipMalloc(&d_rA, nCells * sizeof(double));
    hipMalloc(&d_pA, nCells * sizeof(double));
    hipMalloc(&d_normFactors_tmp, nCells * sizeof(double));
    hipMalloc(&d_wArA_tmp, nCells * sizeof(double));
    hipMalloc(&d_wApA_tmp, nCells * sizeof(double));
    hipMalloc(&reduce_result, sizeof(double));
    // for parallel
    hipMalloc(&scalarSendBufList_, boundary_surface_value_bytes);
    hipMalloc(&scalarRecvBufList_, boundary_surface_value_bytes);

    // preconditioner
    precond_ = new GAMGELLPreconditioner();
    precond_->initialize(GAMGdata_, agglomeration_level);
}

void PCGELLSolver::initGAMGMatrix(const dfMatrixDataBase& dataBase, GAMGStruct *GAMGdata_, int agglomeration_level)
{
    // preconditioner
    precond_->agglomerateMatrix(dataBase, GAMGdata_, agglomeration_level);
}

void PCGELLSolver::solve
(
        const dfMatrixDataBase& dataBase,
        const double* d_internal_coeffs,
        const double* d_boundary_coeffs,
        int* patch_type,
        double* diagPtr,
        double* ellValues,
        int* ellCols,
        int ell_max_count_,
        const double *rhs, 
        double *psi
)
{
    printf("GPU-ELL-PCG::solve start --------------------------------------------\n");

    int nIterations = 0;
 
    const int row_ = dataBase.num_total_cells;
    const int nCells = dataBase.num_cells;

    double wArA = 0.; // TODO: = solverPerf.great_
    double wArAold = wArA;

    // these two int control reduce's scale
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (nCells + threads_per_block - 1) / threads_per_block;

#ifdef PRINT_    
    printf("threads_per_block = %d, blocks_per_grid = %d\n",threads_per_block, blocks_per_grid);
#endif

    double psi_ave = 0.;
    double normFactor = 0.;
    double initialResidual = 0.;
    double finalResidual = 0.;

    // --- reduce psi to get : psi_ave ---
    reduce(nCells, threads_per_block, blocks_per_grid, psi, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&psi_ave, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&psi_ave, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
    psi_ave = psi_ave / row_;

#ifdef PRINT_
    printf("psi_ave = %.10e\n",psi_ave);
#endif

    // --- addInternalCoeffs : diag ---
    // input : d_internal_coeffs
    addInternalCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size, 
        d_internal_coeffs, dataBase.d_boundary_face_cell, diagPtr, patch_type);
    
    // --- SpMV : wA ---
    // input : psi, diag
    SpMV4ELL(dataBase.stream, nCells, diagPtr, ellValues, ellCols, ell_max_count_, psi, d_wA);

#ifdef PARALLEL_
    // --- initMatrixInterfaces & updateMatrixInterfaces : wA ---
    // input : psi (neighbor's psi)
    updateMatrixInterfaces(
        dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        dataBase.neighbProcNo, dataBase.nccl_comm,
        dataBase.interfaceFlag, psi, d_wA, 
        scalarSendBufList_, scalarRecvBufList_,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

    // --- calculate : rA and pA ---
    // input : rhs, wA and diag
    calrAandpA4ELL(dataBase.stream, nCells, d_rA, rhs, d_wA, diagPtr, ellValues, ell_max_count_, d_pA);

        // --- subBoundaryCoeffs : pA ---
    // input : d_boundary_coeffs
    subBoundaryCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, d_pA, patch_type);

    // --- calculate : pA and d_normFactors_tmp ---
    // input : psi_ave and wA, pA, rhs
    calpAandnormFactor(dataBase.stream, nCells, psi_ave, d_pA, d_normFactors_tmp, d_wA, rhs);
    
    // --- reduce d_normFactors_tmp to get : normFactor ---
    reduce(nCells, threads_per_block, blocks_per_grid, d_normFactors_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&normFactor, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&normFactor, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

    normFactor += small_;

#ifdef PRINT_
    printf("normFactor = %.10e\n",normFactor);
#endif

    // --- reduce abs(rA) to get : initialResidual ---
    reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
    hipMemcpyAsync(&initialResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&initialResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
        
    initialResidual = initialResidual / normFactor;

    finalResidual = initialResidual;

#ifdef PRINT_
    printf("first finalResidual = %.10e\n",finalResidual);
#endif

    if
    (
        minIter_ > 0
     || !checkConvergence(finalResidual, initialResidual, nIterations)
    ){

        do{

            wArAold = wArA;

            // TODO: precondition

            // --- calculate : d_wArA_tmp ---
            // input : wA, rA
            AmulBtoC(dataBase.stream, nCells, d_wA, d_rA, d_wArA_tmp);

            // --- reduce d_wArA_tmp to get : wArA ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_wArA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wArA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wArA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

#ifdef PRINT_
            printf("wArA = %.10e\n",wArA);
#endif

            if(nIterations == 0){
                hipMemcpyAsync(d_pA, d_wA, nCells * sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream);
            }
            else{
                double beta = wArA/wArAold;
                // --- calculate : d_pA ---
                // input : wA, beta, d_pA
                calpA(dataBase.stream, nCells, d_pA, d_wA, beta);
            }

            // --- SpMV : wA ---
            // input : pA, diag
            SpMV4ELL(dataBase.stream, nCells, diagPtr, ellValues, ellCols, ell_max_count_, d_pA, d_wA);

#ifdef PARALLEL_
            // --- initMatrixInterfaces & updateMatrixInterfaces wA ---
            // input : pA (neighbor's pA)
            updateMatrixInterfaces(
                dataBase.stream, dataBase.num_patches, dataBase.patch_size,
                dataBase.neighbProcNo, dataBase.nccl_comm,
                dataBase.interfaceFlag, d_pA, d_wA, 
                scalarSendBufList_, scalarRecvBufList_,
                d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

            double wApA = 0.;
            // --- calculate : d_wApA_tmp ---
            // input : wA, pA
            AmulBtoC(dataBase.stream, nCells, d_wA, d_pA, d_wApA_tmp);

            // --- reduce d_wApA_tmp to get : wApA ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_wApA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wApA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wApA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

#ifdef PRINT_
            printf("wApA = %.10e\n",wApA);
#endif

            if (checkSingularity(abs(wApA)/normFactor)) break;

            double alpha = wArA/wApA;
            // --- calculate : psi and d_rA ---
            // input : alpha, d_pA and alpha, d_wA
            calpsiandrA(dataBase.stream, nCells, psi, d_pA, d_rA, d_wA, alpha);

            // --- reduce abs(rA) to get : finalResidual ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
            hipMemcpyAsync(&finalResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&finalResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

            finalResidual = finalResidual / normFactor;

#ifdef PRINT_
            printf("final finalResidual = finalResidual / normFactor : %.10e\n",finalResidual);
#endif
        }while
        (
            (
            ++nIterations < maxIter_
            && !checkConvergence(finalResidual, initialResidual, nIterations)
            )
            || nIterations < minIter_
        );

    }
};


void PCGELLSolver::solve_useGAMG
(
        const dfMatrixDataBase& dataBase,
        const double* d_internal_coeffs,
        const double* d_boundary_coeffs,
        int* patch_type,
        double* diagPtr,
        double* ellValues,
        int* ellCols,
        int ell_max_count_,
        const double *rhs, 
        double *psi,
        GAMGStruct *GAMGdata_, 
        int agglomeration_level
)
{
    printf("GPU-ELL-PCG(GAMG)::solve start --------------------------------------------\n");

    int nIterations = 0;
 
    const int row_ = dataBase.num_total_cells;
    const int nCells = dataBase.num_cells;

    double wArA = 0.; // TODO: = solverPerf.great_
    double wArAold = wArA;

    // these two int control reduce's scale
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (nCells + threads_per_block - 1) / threads_per_block;

#ifdef PRINT_    
    printf("threads_per_block = %d, blocks_per_grid = %d\n",threads_per_block, blocks_per_grid);
#endif

    double psi_ave = 0.;
    double normFactor = 0.;
    double initialResidual = 0.;
    double finalResidual = 0.;

    // --- reduce psi to get : psi_ave ---
    reduce(nCells, threads_per_block, blocks_per_grid, psi, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&psi_ave, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&psi_ave, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
    psi_ave = psi_ave / row_;

#ifdef PRINT_
    printf("psi_ave = %.10e\n",psi_ave);
#endif

    // --- addInternalCoeffs : diag ---
    // input : d_internal_coeffs
    addInternalCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size, 
        d_internal_coeffs, dataBase.d_boundary_face_cell, diagPtr, patch_type);
    
    // --- SpMV : wA ---
    // input : psi, diag
    SpMV4ELL(dataBase.stream, nCells, diagPtr, ellValues, ellCols, ell_max_count_, psi, d_wA);

#ifdef PARALLEL_
    // --- initMatrixInterfaces & updateMatrixInterfaces : wA ---
    // input : psi (neighbor's psi)
    updateMatrixInterfaces(
        dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        dataBase.neighbProcNo, dataBase.nccl_comm,
        dataBase.interfaceFlag, psi, d_wA, 
        scalarSendBufList_, scalarRecvBufList_,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

    // --- calculate : rA and pA ---
    // input : rhs, wA and diag
    calrAandpA4ELL(dataBase.stream, nCells, d_rA, rhs, d_wA, diagPtr, ellValues, ell_max_count_, d_pA);

        // --- subBoundaryCoeffs : pA ---
    // input : d_boundary_coeffs
    subBoundaryCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, d_pA, patch_type);

    // --- calculate : pA and d_normFactors_tmp ---
    // input : psi_ave and wA, pA, rhs
    calpAandnormFactor(dataBase.stream, nCells, psi_ave, d_pA, d_normFactors_tmp, d_wA, rhs);
    
    // --- reduce d_normFactors_tmp to get : normFactor ---
    reduce(nCells, threads_per_block, blocks_per_grid, d_normFactors_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&normFactor, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&normFactor, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

    normFactor += small_;

#ifdef PRINT_
    printf("normFactor = %.10e\n",normFactor);
#endif

    // --- reduce abs(rA) to get : initialResidual ---
    reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
    hipMemcpyAsync(&initialResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&initialResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
        
    initialResidual = initialResidual / normFactor;

    finalResidual = initialResidual;

#ifdef PRINT_
    printf("first finalResidual = %.10e\n",finalResidual);
#endif

    if
    (
        minIter_ > 0
     || !checkConvergence(finalResidual, initialResidual, nIterations)
    ){

        do{

            precond_->precondition(d_wA, d_rA, dataBase, GAMGdata_, agglomeration_level);

            wArAold = wArA;

            // --- calculate : d_wArA_tmp ---
            // input : wA, rA
            AmulBtoC(dataBase.stream, nCells, d_wA, d_rA, d_wArA_tmp);

            // --- reduce d_wArA_tmp to get : wArA ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_wArA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wArA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wArA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

#ifdef PRINT_
            printf("wArA = %.10e\n",wArA);
#endif

            if(nIterations == 0){
                hipMemcpyAsync(d_pA, d_wA, nCells * sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream);
            }
            else{
                double beta = wArA/wArAold;
                // --- calculate : d_pA ---
                // input : wA, beta, d_pA
                calpA(dataBase.stream, nCells, d_pA, d_wA, beta);
            }

            // --- SpMV : wA ---
            // input : pA, diag
            SpMV4ELL(dataBase.stream, nCells, diagPtr, ellValues, ellCols, ell_max_count_, d_pA, d_wA);

#ifdef PARALLEL_
            // --- initMatrixInterfaces & updateMatrixInterfaces wA ---
            // input : pA (neighbor's pA)
            updateMatrixInterfaces(
                dataBase.stream, dataBase.num_patches, dataBase.patch_size,
                dataBase.neighbProcNo, dataBase.nccl_comm,
                dataBase.interfaceFlag, d_pA, d_wA, 
                scalarSendBufList_, scalarRecvBufList_,
                d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

            double wApA = 0.;
            // --- calculate : d_wApA_tmp ---
            // input : wA, pA
            AmulBtoC(dataBase.stream, nCells, d_wA, d_pA, d_wApA_tmp);

            // --- reduce d_wApA_tmp to get : wApA ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_wApA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wApA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wApA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

#ifdef PRINT_
            printf("wApA = %.10e\n",wApA);
#endif

            if (checkSingularity(abs(wApA)/normFactor)) break;

            double alpha = wArA/wApA;
            // --- calculate : psi and d_rA ---
            // input : alpha, d_pA and alpha, d_wA
            calpsiandrA(dataBase.stream, nCells, psi, d_pA, d_rA, d_wA, alpha);

            // --- reduce abs(rA) to get : finalResidual ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
            hipMemcpyAsync(&finalResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&finalResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

            finalResidual = finalResidual / normFactor;

#ifdef PRINT_
            printf("final finalResidual = finalResidual / normFactor : %.10e\n",finalResidual);
#endif
        }while
        (
            (
            ++nIterations < maxIter_
            && !checkConvergence(finalResidual, initialResidual, nIterations)
            )
            || nIterations < minIter_
        );

    }
};
