#include "dfCSRPreconditioner.H"
#include "dfSolverOpBase.H"

// kernel functions for PBiCGStab solver

void GAMGCSRPreconditioner::initialize
(
    GAMGStruct *GAMGdata, int agglomeration_level
)
{
    std::cout << "*** call in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    for(int leveli=0; leveli<agglomeration_level; leveli++)
    {
        std::cout << "   malloc leveli: " << leveli << std::endl;
        // iteration data
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_CorrFields, GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_Sources,    GAMGdata[leveli].nCell*sizeof(double)));

        // temp data for reduce
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_AcfField,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_preSmoothField,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorNum,   GAMGdata[leveli].nCell*sizeof(double)));
        checkCudaErrors(hipMalloc(&GAMGdata[leveli].d_scalingFactorDenom, GAMGdata[leveli].nCell*sizeof(double)));
    }
    std::cout << "*** end in GAMGCSRPreconditioner::initialize(): init Vcycle " << std::endl;
    std::cout << "*********************************************************** " << std::endl;
};

void GAMGCSRPreconditioner::agglomerateMatrix
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::agglomerateMatrix " << std::endl;
    for(int leveli=0; leveli<agglomeration_level-1; leveli++)
    {
        std::cout << "  level: " << leveli << ", in cell: " << GAMGdata_[leveli].nCell
                                           << ", out cell: " << GAMGdata_[leveli+1].nCell << std::endl;

        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_diag, GAMGdata_[leveli+1].d_diag);

        restrictMatrixGPU(dataBase.stream, GAMGdata_[leveli].nFace, 
                        GAMGdata_[leveli].d_faceRestrictMap, GAMGdata_[leveli].d_faceFlipMap,
                        GAMGdata_[leveli].d_upper, GAMGdata_[leveli].d_lower,
                        GAMGdata_[leveli+1].d_upper, GAMGdata_[leveli+1].d_lower, GAMGdata_[leveli+1].d_diag);

        //TODO: calculate interface & interfaceCoef on coarseGrid...
    }
};

void GAMGCSRPreconditioner::fine2coarse
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::fine2coarse " << std::endl;
    for(int leveli=startLevel; leveli<endLevel; leveli++)
    {
        std::cout << "  this level: " << leveli << ", restrict source for coarser level " << std::endl;
        restrictFieldGPU(dataBase.stream, GAMGdata_[leveli].nCell, 
                        GAMGdata_[leveli].d_restrictMap, 
                        GAMGdata_[leveli].d_Sources, GAMGdata_[leveli+1].d_Sources);
        
        checkCudaErrors(hipMemset(GAMGdata_[leveli+1].d_CorrFields, 0, GAMGdata_[leveli+1].nCell*sizeof(double)));

        //TODO: add smoother here to get d_CorrFields for leveli+1 

        if (leveli < endLevel-1)
        {
            //TODO: add scale here for d_CorrFields leveli+1

            //TODO: add Amul to get Acf


            //Purpose: GAMGdata_[leveli+1].d_Sources -= Acf
            updateSourceFieldGPU( dataBase.stream, GAMGdata_[leveli+1].nCell, 
                                  GAMGdata_[leveli+1].d_Sources, GAMGdata_[leveli+1].d_AcfField);
        }    
    }

};

void GAMGCSRPreconditioner::coarse2fine
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level,
    int startLevel, int endLevel
)
{
    std::cout << "********* call in GAMGCSRPreconditioner::coarse2fine " << std::endl;
    for(int leveli=startLevel; leveli>endLevel; leveli--)
    {
        //Purpose: preSmoothedCoarseCorrField = MGCorrFields[leveli-1];
        checkCudaErrors(hipMemcpyAsync(GAMGdata_[leveli-1].d_preSmoothField, GAMGdata_[leveli-1].d_CorrFields, 
                                        GAMGdata_[leveli-1].nCell*sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream));

        std::cout << "  this level: " << leveli << ", prolong correct for finer level " << std::endl;
        prolongFieldGPU(dataBase.stream, GAMGdata_[leveli-1].nCell, 
                        GAMGdata_[leveli-1].d_restrictMap, 
                        GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli].d_CorrFields);

        if (leveli < startLevel - 1)
        {
            //TODO: add scale here for leveli-1
        }
        
        if (leveli > endLevel + 1)
        {
            //Purpose: MGCorrFields[leveli] += preSmoothedCoarseCorrField;
            updateCorrFieldGPU( dataBase.stream, GAMGdata_[leveli-1].nCell, 
                                GAMGdata_[leveli-1].d_CorrFields, GAMGdata_[leveli-1].d_preSmoothField);

            //TODO: add smoother here for leveli-1
        }
    }

};

void GAMGCSRPreconditioner::Vcycle
(
    const dfMatrixDataBase& dataBase,
    GAMGStruct *GAMGdata_, int agglomeration_level
)
{
    bool solveCoarsest = false;

    std::cout << "********* call in GAMGCSRPreconditioner::Vcycle " << std::endl;
    fine2coarse(dataBase, GAMGdata_, agglomeration_level, 0, agglomeration_level-1);

    if (solveCoarsest)
    {
        if (GAMGdata_[agglomeration_level-1].nCell == 1)
        {
            //directSolve1x1
            directSolve1x1GPU(dataBase.stream, GAMGdata_[agglomeration_level-1].nCell, 
                                GAMGdata_[agglomeration_level-1].d_diag, 
                                GAMGdata_[agglomeration_level-1].d_CorrFields, 
                                GAMGdata_[agglomeration_level-1].d_Sources);
        }
        else if (GAMGdata_[agglomeration_level-1].nCell == 4)
        {
            //directSolve4x4
        }
        else
        {
            std::cout << "*** Unsupported dimension for aggregation amg level ..."<< std::endl;
        }
    }

    coarse2fine(dataBase, GAMGdata_, agglomeration_level, agglomeration_level-1, 0);
};

void GAMGCSRPreconditioner::precondition
(
    double *wA,
    const double *rA
)
{
    // Implement the GAMG precondition procedure here
    std::cout << "********* call in GAMGCSRPreconditioner::precondition " << std::endl;
};