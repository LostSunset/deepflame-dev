#include "hip/hip_runtime.h"
#include "dfpEqn.H"
#include "dfSolverOpBase.H"
// #define AMGX_
#define CSR_
// #define ELL_

// #define PARALLEL_

// Start if write vector to file ...
#include <fstream>
#include <iomanip>
#include <iostream>
#include <vector>
#include <string>
#include <nvtx3/nvToolsExt.h>

void readArrayFromBinaryFile(const std::string& filename, double* array, std::size_t size) {
    std::ifstream inFile(filename, std::ios::binary);
    if (!inFile) {
        std::cerr << "Error opening file for reading: " << filename << std::endl;
        return;
    }

    inFile.read(reinterpret_cast<char*>(array), size * sizeof(double));
    inFile.close();
}


__global__ void fvc_interpolate_internal_multi_scalar_kernel(int num_surfaces, const int *lower_index, const int *upper_index,
        const double *vf1, const double *vf2, const double *weight, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;
    
    double w = weight[index];

    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double vf3_owner = vf1[owner] * vf2[owner];
    double vf3_neighbour = vf1[neighbor] * vf2[neighbor];

    output[index] = (w * (vf3_owner - vf3_neighbour) + vf3_neighbour);
}

__global__ void fvc_interpolate_boundary_multi_scalar_kernel_unCouple(int num, int offset,
        const double *boundary_vf1, const double *boundary_vf2, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;
    
    int start_index = offset + index;
    double boundary_vf3 = boundary_vf1[start_index] * boundary_vf2[start_index];
    output[start_index] = boundary_vf3;
}

__global__ void fvc_interpolate_boundary_multi_scalar_kernel_processor(int num, int offset,
        const double *boundary_weight, const double *boundary_vf1, const double *boundary_vf2, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;
    
    int neighbor_start_index = offset + index;
    int internal_start_index = offset + num + index;

    double bouWeight = boundary_weight[neighbor_start_index];

    double neighbor_boundary_vf3 = boundary_vf1[neighbor_start_index] * boundary_vf2[neighbor_start_index];
    double internal_boundary_vf3 = boundary_vf1[internal_start_index] * boundary_vf2[internal_start_index];
    
    double boundary_vf3 = (1 - bouWeight) * neighbor_boundary_vf3 + bouWeight * internal_boundary_vf3;
    
    output[neighbor_start_index] = boundary_vf3;
}

__global__ void get_phiCorr_internal_kernel(int num_cells, int num_surfaces, 
        const int *lower_index, const int *upper_index, const double *phi_old, 
        const double *field_vector, const double *field_scalar, const double *weight, const double *face_vector,
        double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;
    
    double w = weight[index];
    double Sfx = face_vector[num_surfaces * 0 + index];
    double Sfy = face_vector[num_surfaces * 1 + index];
    double Sfz = face_vector[num_surfaces * 2 + index];

    int owner = lower_index[index];
    int neighbor = upper_index[index];

    double vf_own_x = field_vector[num_cells * 0 + owner] * field_scalar[owner];
    double vf_own_y = field_vector[num_cells * 1 + owner] * field_scalar[owner];
    double vf_own_z = field_vector[num_cells * 2 + owner] * field_scalar[owner];

    double vf_nei_x = field_vector[num_cells * 0 + neighbor] * field_scalar[neighbor];
    double vf_nei_y = field_vector[num_cells * 1 + neighbor] * field_scalar[neighbor];
    double vf_nei_z = field_vector[num_cells * 2 + neighbor] * field_scalar[neighbor];

    double ssfx = (w * (vf_own_x - vf_nei_x) + vf_nei_x);
    double ssfy = (w * (vf_own_y - vf_nei_y) + vf_nei_y);
    double ssfz = (w * (vf_own_z - vf_nei_z) + vf_nei_z);

    output[index] = phi_old[index] - (Sfx * ssfx + Sfy * ssfy + Sfz * ssfz);    
}

__global__ void get_phiCorr_boundary_kernel_zeroGradient(int num_boundary_surfaces, int num, int offset,
        const double *boundary_face_vector, const double *boundary_field_vector, 
        const double *boundary_field_scalar, const double *boundary_phi_old, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;
    
    int start_index = offset + index;

    double bouSfx = boundary_face_vector[num_boundary_surfaces * 0 + start_index];
    double bouSfy = boundary_face_vector[num_boundary_surfaces * 1 + start_index];
    double bouSfz = boundary_face_vector[num_boundary_surfaces * 2 + start_index];

    double boussfx = boundary_field_vector[num_boundary_surfaces * 0 + start_index] * boundary_field_scalar[start_index];
    double boussfy = boundary_field_vector[num_boundary_surfaces * 1 + start_index] * boundary_field_scalar[start_index];
    double boussfz = boundary_field_vector[num_boundary_surfaces * 2 + start_index] * boundary_field_scalar[start_index];

    output[start_index] = boundary_phi_old[start_index] - (bouSfx * boussfx + bouSfy * boussfy + bouSfz * boussfz);
}

__global__ void get_phiCorr_boundary_kernel_processor(int num_boundary_surfaces, int num, int offset,
        const double *boundary_face_vector, const double *boundary_field_vector, 
        const double *boundary_field_scalar, const double *boundary_phi_old, 
        const double *boundary_weight, double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;
    
    int neighbor_start_index = offset + index;
    int internal_start_index = offset + num + index;

    double bouWeight = boundary_weight[neighbor_start_index];

    double bouSfx = boundary_face_vector[num_boundary_surfaces * 0 + neighbor_start_index];
    double bouSfy = boundary_face_vector[num_boundary_surfaces * 1 + neighbor_start_index];
    double bouSfz = boundary_face_vector[num_boundary_surfaces * 2 + neighbor_start_index];

    double boussfxNeighbor = boundary_field_vector[num_boundary_surfaces * 0 + neighbor_start_index] 
            * boundary_field_scalar[neighbor_start_index];
    double boussfyNeighbor = boundary_field_vector[num_boundary_surfaces * 1 + neighbor_start_index] 
            * boundary_field_scalar[neighbor_start_index];
    double boussfzNeighbor = boundary_field_vector[num_boundary_surfaces * 2 + neighbor_start_index] 
            * boundary_field_scalar[neighbor_start_index];
    
    double boussfxInternal = boundary_field_vector[num_boundary_surfaces * 0 + internal_start_index] 
            * boundary_field_scalar[internal_start_index];
    double boussfyInternal = boundary_field_vector[num_boundary_surfaces * 1 + internal_start_index] 
            * boundary_field_scalar[internal_start_index];
    double boussfzInternal = boundary_field_vector[num_boundary_surfaces * 2 + internal_start_index] 
            * boundary_field_scalar[internal_start_index];
    
    double boussfx = (1 - bouWeight) * boussfxNeighbor + bouWeight * boussfxInternal;
    double boussfy = (1 - bouWeight) * boussfyNeighbor + bouWeight * boussfyInternal;
    double boussfz = (1 - bouWeight) * boussfzNeighbor + bouWeight * boussfzInternal;

    output[neighbor_start_index] = boundary_phi_old[neighbor_start_index] - (bouSfx * boussfx + bouSfy * boussfy + bouSfz * boussfz);
}

__global__ void get_ddtCorr_internal_kernel(int num_cells, int num_surfaces, 
        const double *phiCorr, const double *phi, const double rDeltaT,
        double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;
    
    double phiCorrVal = phiCorr[index];
    double phiVal = phi[index];

    double tddtCouplingCoeff = 1. - min(fabs(phiCorrVal)/fabs(phiVal) + SMALL, 1.);
    
    output[index] = tddtCouplingCoeff * rDeltaT * phiCorrVal;
}

__global__ void get_ddtCorr_boundary_nonZero_kernel(int num_boundary_surfaces, int num, int offset,
        const double *boundary_phiCorr, const double *boundary_phi, const double rDeltaT,
        double *output)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;
    
    int start_index = offset + index;

    double bouPhiCorrVal = boundary_phiCorr[start_index];
    double bouPhiVal = boundary_phi[start_index];

    double bou_tddtCouplingCoeff = 1. - min(fabs(bouPhiCorrVal)/fabs(bouPhiVal) + SMALL, 1.);
    output[start_index] = bou_tddtCouplingCoeff * rDeltaT * bouPhiCorrVal;
}

__global__ void multi_fvc_flux_fvc_intepolate_internal_kernel(int num_cells, int num_surfaces, 
        const int *lower_index, const int *upper_index,
        const double *field_vector, const double *vf, const double *weight, const double *face_vector,
        double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;
    
    double w = weight[index];
    int owner = lower_index[index];
    int neighbor = upper_index[index];

    // fvc_flux_HbyA
    double Sfx = face_vector[num_surfaces * 0 + index];
    double Sfy = face_vector[num_surfaces * 1 + index];
    double Sfz = face_vector[num_surfaces * 2 + index];

    double ssfx = (w * (field_vector[num_cells * 0 + owner] - field_vector[num_cells * 0 + neighbor]) + field_vector[num_cells * 0 + neighbor]);
    double ssfy = (w * (field_vector[num_cells * 1 + owner] - field_vector[num_cells * 1 + neighbor]) + field_vector[num_cells * 1 + neighbor]);
    double ssfz = (w * (field_vector[num_cells * 2 + owner] - field_vector[num_cells * 2 + neighbor]) + field_vector[num_cells * 2 + neighbor]);

    // fvc_interpolate_rho
    double vf_interp = (w * (vf[owner] - vf[neighbor]) + vf[neighbor]);

    output[index] += (Sfx * ssfx + Sfy * ssfy + Sfz * ssfz) * vf_interp;
}

__global__ void multi_fvc_flux_fvc_intepolate_boundary_kernel_zeroGradient(int num_boundary_surfaces, int num, int offset, 
        const double *boundary_face_vector, const double *boundary_field_vector, 
        const double *boundary_vf, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;
    
    int start_index = offset + index;

    double bouSfx = boundary_face_vector[num_boundary_surfaces * 0 + start_index];
    double bouSfy = boundary_face_vector[num_boundary_surfaces * 1 + start_index];
    double bouSfz = boundary_face_vector[num_boundary_surfaces * 2 + start_index];

    double boussfx = boundary_field_vector[num_boundary_surfaces * 0 + start_index];
    double boussfy = boundary_field_vector[num_boundary_surfaces * 1 + start_index];
    double boussfz = boundary_field_vector[num_boundary_surfaces * 2 + start_index];

    output[start_index] += (bouSfx * boussfx + bouSfy * boussfy + bouSfz * boussfz) * boundary_vf[start_index];
}

__global__ void multi_fvc_flux_fvc_intepolate_boundary_kernel_processor(int num_boundary_surfaces, int num, int offset, 
        const double *boundary_face_vector, const double *boundary_field_vector, const double *boundary_weight,
        const double *boundary_vf, double *output, double sign)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num)
        return;
    
    int neighbor_start_index = offset + index;
    int internal_start_index = offset + num + index;

    double bouWeight = boundary_weight[neighbor_start_index];

    double bouSfx = boundary_face_vector[num_boundary_surfaces * 0 + neighbor_start_index];
    double bouSfy = boundary_face_vector[num_boundary_surfaces * 1 + neighbor_start_index];
    double bouSfz = boundary_face_vector[num_boundary_surfaces * 2 + neighbor_start_index];

    // interpolate boundary vector
    double boussfx = (1 - bouWeight) * boundary_field_vector[num_boundary_surfaces * 0 + neighbor_start_index] + 
            bouWeight * boundary_field_vector[num_boundary_surfaces * 0 + internal_start_index];
    double boussfy = (1 - bouWeight) * boundary_field_vector[num_boundary_surfaces * 1 + neighbor_start_index] + 
            bouWeight * boundary_field_vector[num_boundary_surfaces * 1 + internal_start_index];
    double boussfz = (1 - bouWeight) * boundary_field_vector[num_boundary_surfaces * 2 + neighbor_start_index] + 
            bouWeight * boundary_field_vector[num_boundary_surfaces * 2 + internal_start_index];
    
    // interpolate boundary scalar
    double bouvf = (1 - bouWeight) * boundary_vf[neighbor_start_index] + bouWeight * boundary_vf[internal_start_index];
    
    output[neighbor_start_index] += (bouSfx * boussfx + bouSfy * boussfy + bouSfz * boussfz) * bouvf;
}

__global__ void correct_diag_mtx_multi_tpsi_kernel(int num_cells, const double *psi, const double *thermo_psi, 
        double *source, double *diag)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_cells)
        return;

    // correction: source += (-diag * psi + source)
    double srcVal = source[index];
    double APsi = - diag[index] * psi[index] + srcVal;
    source[index] -= APsi;

    // multi psi
    double tPsiVal = thermo_psi[index];
    source[index] *= tPsiVal;
    diag[index] *= tPsiVal;
}

double* dfpEqn::getFieldPointer(const char* fieldAlias, location loc, position pos) {
    char mergedName[256];
    if (pos == position::internal) {
        sprintf(mergedName, "%s_%s", (loc == location::cpu) ? "h" : "d", fieldAlias);
    } else if (pos == position::boundary) {
        sprintf(mergedName, "%s_boundary_%s", (loc == location::cpu) ? "h" : "d", fieldAlias);
    }

    double *pointer = nullptr;
    if (fieldPointerMap.find(std::string(mergedName)) != fieldPointerMap.end()) {
        pointer = fieldPointerMap[std::string(mergedName)];
    }
    if (pointer == nullptr) {
        fprintf(stderr, "Warning! getFieldPointer of %s returns nullptr!\n", mergedName);
    }
    //fprintf(stderr, "fieldAlias: %s, mergedName: %s, pointer: %p\n", fieldAlias, mergedName, pointer);

    return pointer;
}

void dfpEqn::setConstantValues(const std::string &mode_string, const std::string &setting_path,
                GAMGStruct *GAMGdata_, int agglomeration_level) {
    this->stream = dataBase_.stream;
    this->mode_string = mode_string;
    this->setting_path = setting_path;
#ifdef AMGX_
    pSolver = new AmgXSolver(mode_string, setting_path, dataBase_.localRank);
#endif

#ifdef CSR_
    bool useGAMG = true;
    pCSRSolver = new PCGCSRSolver();
    pCSRSolver->initSolvePerformance
    (     
        1e-20, //small_
        2.22507e-308, //vsmall_
        4, //maxIter_ 
        4, //minIter_ 
        1e-9, //tolerance_ 
        0.01 //relTol_
    );
    if (useGAMG)
    {
        pCSRSolver->initializeGAMG(dataBase_, dataBase_.num_cells, dataBase_.boundary_surface_value_bytes,
                            GAMGdata_, agglomeration_level);
    }
    else 
    {
        pCSRSolver->initialize(dataBase_.num_cells, dataBase_.boundary_surface_value_bytes);
    }
#endif

#ifdef ELL_
    bool useGAMG = true;
    pELLSolver = new PCGELLSolver();
    pELLSolver->initSolvePerformance
    (     
        1e-20, //small_
        2.22507e-308, //vsmall_
        4, //maxIter_ 
        4, //minIter_ 
        1e-9, //tolerance_ 
        0.01 //relTol_
    );
    if (useGAMG)
    {
        pELLSolver->initializeGAMG(dataBase_, dataBase_.num_cells, dataBase_.boundary_surface_value_bytes,
                            GAMGdata_, agglomeration_level);
    }
    else 
    {
        pELLSolver->initialize(dataBase_.num_cells, dataBase_.boundary_surface_value_bytes);
    }
#endif
}

void dfpEqn::setConstantFields(const std::vector<int> patch_type_U, const std::vector<int> patch_type_p) {
    this->patch_type_U = patch_type_U;
    this->patch_type_p = patch_type_p;
}

void dfpEqn::createNonConstantFieldsInternal() {
#ifndef STREAM_ALLOCATOR
    // intermediate fields
    checkCudaErrors(hipMalloc((void**)&d_rhorAUf, dataBase_.surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_phiHbyA, dataBase_.surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_flux, dataBase_.surface_value_bytes));
#endif
}

void dfpEqn::createNonConstantFieldsBoundary() {
#ifndef STREAM_ALLOCATOR
    // boundary coeffs
    checkCudaErrors(hipMalloc((void**)&d_value_internal_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_value_boundary_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_gradient_internal_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_gradient_boundary_coeffs, dataBase_.boundary_surface_value_bytes));
    // intermediate boundary fields
    checkCudaErrors(hipMalloc((void**)&d_boundary_rhorAUf, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_boundary_phiHbyA, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_boundary_flux, dataBase_.boundary_surface_value_bytes));
#endif
}

void dfpEqn::createNonConstantLduAndCsrFields() {
    // ldu and csr
    checkCudaErrors(hipMalloc((void**)&d_ldu, dataBase_.csr_value_bytes));
#ifdef AMGX_
    d_lower = d_ldu;
    d_diag = d_ldu + dataBase_.num_surfaces;
    d_upper = d_ldu + dataBase_.num_cells + dataBase_.num_surfaces;
#endif
#ifdef CSR_
    d_lower = d_ldu;
    d_upper = d_ldu + dataBase_.num_surfaces;
    d_diag = d_ldu + 2 * dataBase_.num_surfaces;
#endif
#ifdef ELL_
    d_lower = d_ldu;
    d_upper = d_ldu + dataBase_.num_surfaces;
    d_diag = d_ldu + 2 * dataBase_.num_surfaces;
#endif
    d_extern = d_ldu + dataBase_.num_cells + 2 * dataBase_.num_surfaces;
#ifndef STREAM_ALLOCATOR
    checkCudaErrors(hipMalloc((void**)&d_source, dataBase_.cell_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_internal_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_boundary_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_A, dataBase_.csr_value_bytes));
#endif
}

void dfpEqn::GAMGpEqnldu(GAMGStruct *GAMGdata_, int agglomeration_level, int type){
    GAMGlduPtr.resize(agglomeration_level);
    //csr
    if(type ==1){
        GAMGldu2csrPtr.resize(agglomeration_level);
        for(int i = 0; i < agglomeration_level; i++){
            checkCudaErrors(hipMalloc((void**)&GAMGlduPtr[i], GAMGdata_[i].nFace * 2 * sizeof(double)));
            checkCudaErrors(hipMalloc((void**)&GAMGldu2csrPtr[i], GAMGdata_[i].nFace * 2 * sizeof(int)));
            checkCudaErrors(hipMemcpy(GAMGldu2csrPtr[i], dataBase_.h_ldu_to_csr_no_diag[i], GAMGdata_[i].nFace * 2 * sizeof(int), hipMemcpyHostToDevice));
        }
    }
    //ell
    if(type==2){
        GAMGldu2ellPtr.resize(agglomeration_level);
        for(int i = 0; i < agglomeration_level; i++){
            checkCudaErrors(hipMalloc((void**)&GAMGlduPtr[i], GAMGdata_[i].nFace * 2 * sizeof(double)));
            checkCudaErrors(hipMalloc((void**)&GAMGldu2ellPtr[i], GAMGdata_[i].nFace * 2 * sizeof(int)));
            checkCudaErrors(hipMemcpy(GAMGldu2ellPtr[i], dataBase_.h_ldu2ellIndex[i], GAMGdata_[i].nFace * 2 * sizeof(int), hipMemcpyHostToDevice));
        }
    }

}

void dfpEqn::initNonConstantFields(const double *p, const double *boundary_p){
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_p, dataBase_.h_p, dataBase_.cell_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_boundary_p, dataBase_.h_boundary_p, dataBase_.boundary_surface_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
}

void dfpEqn::cleanCudaResources() {
#ifdef USE_GRAPH
    if (pre_graph_created) {
        checkCudaErrors(hipGraphExecDestroy(graph_instance_pre));
        checkCudaErrors(hipGraphDestroy(graph_pre));
    }
    if (post_graph_created) {
        checkCudaErrors(hipGraphExecDestroy(graph_instance_post));
        checkCudaErrors(hipGraphDestroy(graph_post));
    }
#endif
}

// tmp
void dfpEqn::preProcess(double *h_phi, double *h_boundary_phi) {
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_phi, h_phi, dataBase_.surface_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_boundary_phi, h_boundary_phi, dataBase_.boundary_surface_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
};

void dfpEqn::correctPsi(const double *h_thermoPsi, double *h_boundary_thermoPsi) {
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_thermo_psi, h_thermoPsi, dataBase_.cell_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_boundary_thermo_psi, h_boundary_thermoPsi, dataBase_.boundary_surface_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
};
void dfpEqn::correctP(const double *h_p, double *h_boundary_p) {
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_p, h_p, dataBase_.cell_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_boundary_p, h_boundary_p, dataBase_.boundary_surface_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
};

void dfpEqn::process(GAMGStruct *GAMGdata, int agglomeration_level) {
    TICK_INIT_EVENT;
    TICK_START_EVENT;
#ifdef USE_GRAPH
    if(!pre_graph_created) {
        DEBUG_TRACE;
        checkCudaErrors(hipStreamBeginCapture(dataBase_.stream, hipStreamCaptureModeGlobal));
#endif

#ifdef STREAM_ALLOCATOR
    // intermediate fields
    checkCudaErrors(hipMallocAsync((void**)&d_rhorAUf, dataBase_.surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_phiHbyA, dataBase_.surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_flux, dataBase_.surface_value_bytes, dataBase_.stream));

    // boundary coeffs
    checkCudaErrors(hipMallocAsync((void**)&d_value_internal_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_value_boundary_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_gradient_internal_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_gradient_boundary_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    // intermediate boundary fields
    checkCudaErrors(hipMallocAsync((void**)&d_boundary_rhorAUf, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_boundary_phiHbyA, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_boundary_flux, dataBase_.boundary_surface_value_bytes, dataBase_.stream));

    // ldu and csr
    checkCudaErrors(hipMallocAsync((void**)&d_source, dataBase_.cell_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_internal_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_boundary_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_A, dataBase_.csr_value_bytes, dataBase_.stream));
#endif

    checkCudaErrors(hipMemsetAsync(d_ldu, 0, dataBase_.csr_value_bytes, dataBase_.stream)); // d_ldu contains d_lower, d_diag, and d_upper
    checkCudaErrors(hipMemsetAsync(d_source, 0, dataBase_.cell_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_internal_coeffs, 0, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_boundary_coeffs, 0, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_A, 0, dataBase_.csr_value_bytes, dataBase_.stream));

    // intermediate parameters
    checkCudaErrors(hipMemsetAsync(d_rhorAUf, 0, dataBase_.surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_boundary_rhorAUf, 0, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_phiHbyA, 0, dataBase_.surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_boundary_phiHbyA, 0, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_flux, 0, dataBase_.surface_value_bytes, dataBase_.stream)); // TODO: introduce of flux is not necessary
    
    update_boundary_coeffs_scalar(dataBase_.stream,
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_p.data(),
            dataBase_.d_boundary_delta_coeffs, dataBase_.d_boundary_p, dataBase_.d_boundary_weight,
            d_value_internal_coeffs, d_value_boundary_coeffs,
            d_gradient_internal_coeffs, d_gradient_boundary_coeffs);
    getrhorAUf(dataBase_.stream, dataBase_.num_cells, dataBase_.num_surfaces, 
            dataBase_.d_owner, dataBase_.d_neighbor, dataBase_.d_weight, 
            dataBase_.d_rho, dataBase_.d_rAU, d_rhorAUf, // end for internal
            dataBase_.num_patches, dataBase_.patch_size.data(), dataBase_.patch_type_calculated.data(), dataBase_.d_boundary_weight,
            dataBase_.d_boundary_rho, dataBase_.d_boundary_rAU, d_boundary_rhorAUf);
    getphiHbyA(dataBase_.stream, dataBase_.num_cells, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces, 
            dataBase_.rdelta_t, dataBase_.d_owner, dataBase_.d_neighbor, 
            dataBase_.d_weight, dataBase_.d_u_old, dataBase_.d_rho_old,
            dataBase_.d_phi_old, dataBase_.d_rho, d_rhorAUf, dataBase_.d_HbyA, dataBase_.d_sf, d_phiHbyA, // end for internal
            dataBase_.num_patches, dataBase_.patch_size.data(), dataBase_.patch_type_extropolated.data(),
            dataBase_.d_boundary_sf, dataBase_.d_boundary_u_old, dataBase_.d_boundary_rho, 
            dataBase_.d_boundary_rho_old, dataBase_.d_boundary_phi_old, d_boundary_rhorAUf, dataBase_.d_boundary_HbyA, 
            dataBase_.d_boundary_weight, d_boundary_phiHbyA, 1.0);
    fvm_ddt_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.rdelta_t, dataBase_.d_p_old, dataBase_.d_volume, d_diag, d_source);
    correctionDiagMtxMultiTPsi(dataBase_.stream, dataBase_.num_cells, dataBase_.d_p, dataBase_.d_thermo_psi, d_diag, d_source);
    fvc_ddt_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.rdelta_t, dataBase_.d_rho, dataBase_.d_rho_old, dataBase_.d_volume, 
            d_source, -1.);
    fvc_div_surface_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces,
            dataBase_.d_owner, dataBase_.d_neighbor, d_phiHbyA, dataBase_.d_boundary_face_cell, 
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_p.data(),
            d_boundary_phiHbyA, dataBase_.d_volume, d_source, -1.);
    fvm_laplacian_surface_scalar_vol_scalar(dataBase_.stream, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces,
            dataBase_.d_owner, dataBase_.d_neighbor, dataBase_.d_mag_sf, dataBase_.d_delta_coeffs, d_rhorAUf, 
            d_lower, d_upper, d_diag, dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_p.data(),
            dataBase_.d_boundary_mag_sf, d_boundary_rhorAUf, d_gradient_internal_coeffs, d_gradient_boundary_coeffs, 
            d_internal_coeffs, d_boundary_coeffs, -1.);
    
#ifdef AMGX_
    nvtxRangePushA("AMGX::ldutocsr()");
    ldu_to_csr_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces,
            dataBase_.num_Nz, dataBase_.d_boundary_face_cell, dataBase_.d_ldu_to_csr_index,
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_p.data(),
            d_ldu, d_source, d_internal_coeffs, d_boundary_coeffs, d_A);
    nvtxRangePop();
#endif

    // // load CPU data for test ---start
    // std::string abPath = "/root/gpuversion0417/tmp0418/test0514/cvodeIntegrator_64/";
    // int mpirank;
    // MPI_Comm_rank(MPI_COMM_WORLD, &mpirank);
    // std::string c = std::to_string(mpirank);        
    // std::string sdiag = "diag";
    // std::string supper = "upper";
    // std::string slower = "lower";
    // std::string ssource = "source";
    // std::string sIntCoef = "IntCoef";
    // std::string sBouCoef = "BouCoef";
    // std::string b = ".bin";
    
    // bool loadCPUdata4test = true;
    // if (loadCPUdata4test){
    //     printf("*** LOADing data from files ***\n");
    //     // std::string filename_diag = "/root/gpuversion0417/tmp0418/test0514/cvodeIntegrator_64/diag.bin";
    //     // std::string filename_upper = "/root/gpuversion0417/tmp0418/test0514/cvodeIntegrator_64/upper.bin";
    //     // std::string filename_lower = "/root/gpuversion0417/tmp0418/test0514/cvodeIntegrator_64/lower.bin";
    //     // std::string filename_source = "/root/gpuversion0417/tmp0418/test0514/cvodeIntegrator_64/source.bin";
    //     std::string filename_diag = abPath + sdiag + c + b;
    //     std::string filename_upper = abPath + supper + c + b;
    //     std::string filename_lower = abPath + slower + c + b;
    //     std::string filename_source = abPath + ssource + c + b;
    //     std::string filename_IntCoef = abPath + sIntCoef + c + b;
    //     std::string filename_BouCoef = abPath + sBouCoef + c + b;
    //     double* data_diag;
    //     double* data_upper;
    //     double* data_lower;
    //     double* data_source;
    //     double* data_IntCoef;
    //     double* data_BouCoef;
    //     data_diag = (double*)malloc(sizeof(double)*dataBase_.num_cells);
    //     data_upper = (double*)malloc(sizeof(double)*dataBase_.num_surfaces);
    //     data_lower = (double*)malloc(sizeof(double)*dataBase_.num_surfaces);
    //     data_source = (double*)malloc(sizeof(double)*dataBase_.num_cells);
    //     data_IntCoef = (double*)malloc(sizeof(double)*dataBase_.num_boundary_surfaces);
    //     data_BouCoef = (double*)malloc(sizeof(double)*dataBase_.num_boundary_surfaces);;
    //     readArrayFromBinaryFile(filename_diag, data_diag, dataBase_.num_cells);
    //     readArrayFromBinaryFile(filename_upper, data_upper, dataBase_.num_surfaces);
    //     readArrayFromBinaryFile(filename_lower, data_lower, dataBase_.num_surfaces);
    //     readArrayFromBinaryFile(filename_source, data_source, dataBase_.num_cells);
    //     readArrayFromBinaryFile(filename_IntCoef, data_IntCoef, dataBase_.num_boundary_surfaces);
    //     readArrayFromBinaryFile(filename_BouCoef, data_BouCoef, dataBase_.num_boundary_surfaces);

    //     hipMemcpy(d_diag,  &data_diag[0],  sizeof(double)*dataBase_.num_cells,    hipMemcpyHostToDevice);
    //     hipMemcpy(d_lower, &data_lower[0], sizeof(double)*dataBase_.num_surfaces, hipMemcpyHostToDevice);
    //     hipMemcpy(d_upper, &data_upper[0], sizeof(double)*dataBase_.num_surfaces, hipMemcpyHostToDevice);
    //     hipMemcpy(d_source, &data_source[0], sizeof(double)*dataBase_.num_cells, hipMemcpyHostToDevice);
    //     hipMemcpy(d_internal_coeffs, &data_IntCoef[0], sizeof(double)*dataBase_.num_boundary_surfaces, hipMemcpyHostToDevice);
    //     hipMemcpy(d_boundary_coeffs, &data_BouCoef[0], sizeof(double)*dataBase_.num_boundary_surfaces, hipMemcpyHostToDevice);
    // }

#ifdef CSR_

        // ========================================================================================
        // startif use GAMG ... save for finest matrix
        std::cout << "=== start agglomeration in dfpEqn::process() " << std::endl;
        std::cout << "dataBase_.num_cells: " << dataBase_.num_cells 
                  << ", dataBase_.num_surfaces: " << dataBase_.num_surfaces
                  << ", dataBase_.num_boundary_surfaces: " << dataBase_.num_boundary_surfaces << std::endl;

        // malloc & memset coarse level
        nvtxRangePushA("Memset CSR Matrix");
        for(int leveli=0; leveli<agglomeration_level; leveli++)
        {
            std::cout << "malloc and memset leveli: " << leveli << std::endl;
            // Set matrix data
            if(leveli==0)
            {

                // --- addInternalCoeffs : diag ---
                // input : d_internal_coeffs
                addInternalCoeffs(dataBase_.stream, dataBase_.num_patches, dataBase_.patch_size, 
                    d_internal_coeffs, dataBase_.d_boundary_face_cell, d_diag, dataBase_.patch_type_extropolated.data());
                
                checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_lower, d_lower, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream));
                checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_upper, d_upper, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream));
                checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_diag, d_diag, GAMGdata[leveli].nCell * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_off_diag_value, 0, GAMGdata[leveli].nFace * 2 * sizeof(double)));
                // checkCudaErrors(hipMemset(GAMGdata[leveli].d_csr_row_index_no_diag, 0, (GAMGdata[leveli].nCell + 1) * sizeof(int)));
                // checkCudaErrors(hipMemset(GAMGdata[leveli].d_csr_col_index_no_diag, 0, GAMGdata[leveli].nFace * 2 * sizeof(int)));
            }
            else
            {
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_lower, 0, GAMGdata[leveli].nFace * sizeof(double)));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_upper, 0, GAMGdata[leveli].nFace * sizeof(double)));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_diag, 0, GAMGdata[leveli].nCell * sizeof(double)));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_off_diag_value, 0, GAMGdata[leveli].nFace * 2 * sizeof(double)));
                // checkCudaErrors(hipMemset(GAMGdata[leveli].d_csr_row_index_no_diag, 0, (GAMGdata[leveli].nCell + 1) * sizeof(int)));
                // checkCudaErrors(hipMemset(GAMGdata[leveli].d_csr_col_index_no_diag, 0, GAMGdata[leveli].nFace * 2 * sizeof(int)));
            }


#ifdef PARALLEL_
            if (leveli==0)
            {
                getInterfacesCoeffs(dataBase_.stream, dataBase_.num_patches, dataBase_.patch_size, 
                                    dataBase_.interfaceFlag, dataBase_.patch_type_extropolated.data(), 
                                    d_boundary_coeffs, d_internal_coeffs, 
                                    GAMGdata[0].d_interfaceBouCoeffs, GAMGdata[0].d_interfaceIntCoeffs);

                bool loadCPUCoeffdata4test = false;
                if (loadCPUCoeffdata4test){
                    int mpirank = dataBase_.localRank;
                    for(int patchi=0; patchi<GAMGdata[leveli].nPatchFaces.size(); patchi++)
                    {
                        if (GAMGdata[leveli].nPatchFaces[patchi] > 0)
                        {
                            std::stringstream filenameBouCoeff; 
                            std::stringstream filenameIntCoeff; 

                            filenameBouCoeff << "/root/0427/deepflame-dev/examples/dfLowMachFoam/cvodeIntegrator_64/h_bouCoeff_0_" << patchi << "_" << mpirank << "_Ref.txt";
                            filenameIntCoeff << "/root/0427/deepflame-dev/examples/dfLowMachFoam/cvodeIntegrator_64/h_intCoeff_0_" << patchi << "_" << mpirank << "_Ref.txt";

                            std::string filename_bouCoeff = filenameBouCoeff.str();
                            std::string filename_intCoeff = filenameIntCoeff.str();

                            std::vector<double> data_bouCoeff, data_intCoeff;

                            int offset = 0;
                            std::ifstream file_bouCoeff(filename_bouCoeff);
                            if (!file_bouCoeff.is_open()) {
                                    std::cerr << "error open filename_bouCoeff : " << filename_bouCoeff << std::endl;
                            }
                            double number_bouCoeff;
                            while (file_bouCoeff >> std::setprecision(10) >> number_bouCoeff){
                                if(offset < GAMGdata[leveli].nPatchFaces[patchi]){
                                    data_bouCoeff.push_back(number_bouCoeff);
                                }
                                offset++;
                            }
                            file_bouCoeff.close();

                            offset = 0;
                            std::ifstream file_intCoeff(filename_intCoeff);
                            if (!file_intCoeff.is_open()) {
                                    std::cerr << "error open filename_intCoeff : " << filename_intCoeff << std::endl;
                            }
                            double number_intCoeff;
                            while (file_intCoeff >> std::setprecision(10) >> number_intCoeff){
                                if(offset < GAMGdata[leveli].nPatchFaces[patchi]){
                                    data_intCoeff.push_back(number_intCoeff);
                                }
                                offset++;
                            }
                            file_intCoeff.close();

                            hipMemcpy(GAMGdata[0].d_interfaceBouCoeffs[patchi], &data_bouCoeff[0], sizeof(double)*GAMGdata[leveli].nPatchFaces[patchi], hipMemcpyHostToDevice);
                            hipMemcpy(GAMGdata[0].d_interfaceIntCoeffs[patchi], &data_intCoeff[0], sizeof(double)*GAMGdata[leveli].nPatchFaces[patchi], hipMemcpyHostToDevice);
                        }
                    }
                }
            }
            else
            {
                // Set interface coef data
                for(int patchi=0; patchi<GAMGdata[leveli].nPatchFaces.size(); patchi++)
                {
                    if (GAMGdata[leveli].nPatchFaces[patchi] > 0)
                    {
                        checkCudaErrors(hipMemset(GAMGdata[leveli].d_interfaceBouCoeffs[patchi], 0, GAMGdata[leveli].nPatchFaces[patchi]*sizeof(double)));
                        checkCudaErrors(hipMemset(GAMGdata[leveli].d_interfaceIntCoeffs[patchi], 0, GAMGdata[leveli].nPatchFaces[patchi]*sizeof(double)));
                    
                    }
                }
            }
#endif
        }
        nvtxRangePop();

        std::cout << "=============================================" << std::endl;
        std::cout << "====== call pCSRSolver->initGAMGMatrix ======" << std::endl;
        nvtxRangePushA("initGAMGMatrix()");
        pCSRSolver->initGAMGMatrix(dataBase_, GAMGdata, agglomeration_level);
        nvtxRangePop();
        std::cout << "====== end pCSRSolver->initGAMGMatrix =======" << std::endl;
        std::cout << "=============================================" << std::endl;

        // coarse level ldu to csr
        nvtxRangePushA("ldu to csr");
        for(int leveli=0; leveli<agglomeration_level; leveli++)
        {
            // checkCudaErrors(hipMemcpy(GAMGlduPtr[leveli], GAMGdata[leveli].d_lower, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice));
            // checkCudaErrors(hipMemcpy(GAMGlduPtr[leveli] + GAMGdata[leveli].nFace, GAMGdata[leveli].d_upper, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice));
            
            nvtxRangePushA("peqn_ldu_to_csr_no_diag()");
            // peqn_ldu_to_csr_no_diag(GAMGdata[leveli].nFace * 2, GAMGldu2csrPtr[leveli], GAMGlduPtr[leveli], GAMGdata[leveli].d_off_diag_value);
            peqn_ldu_to_csr_no_diag(GAMGdata[leveli].nFace * 2, GAMGldu2csrPtr[leveli], GAMGdata[leveli].d_lower, GAMGdata[leveli].d_upper, GAMGdata[leveli].d_off_diag_value);
            nvtxRangePop();

            // checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_lowerAddr, &GAMGdata[leveli].lowerAddr[0], GAMGdata[leveli].nFace * sizeof(int), hipMemcpyHostToDevice, dataBase_.stream));
            // checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_upperAddr, &GAMGdata[leveli].upperAddr[0], GAMGdata[leveli].nFace * sizeof(int), hipMemcpyHostToDevice, dataBase_.stream));
            // peqn_ldu2csr(dataBase_.stream, GAMGdata[leveli].nCell, GAMGdata[leveli].nFace, GAMGdata[leveli].d_lower, GAMGdata[leveli].d_upper, GAMGdata[leveli].d_lowerAddr, GAMGdata[leveli].d_upperAddr, 
            //     GAMGdata[leveli].d_off_diag_value, GAMGdata[leveli].d_csr_row_index_no_diag, GAMGdata[leveli].d_csr_col_index_no_diag);
        }
        nvtxRangePop();
#endif

#ifdef ELL_
        // ========================================================================================
        // startif use GAMG ... save for finest matrix
        std::cout << "=== start agglomeration in dfpEqn::process() " << std::endl;
        std::cout << "dataBase_.num_cells: " << dataBase_.num_cells 
                  << ", dataBase_.num_surfaces: " << dataBase_.num_surfaces
                  << ", dataBase_.num_boundary_surfaces: " << dataBase_.num_boundary_surfaces << std::endl;

        // malloc & memset coarse level
        nvtxRangePushA("Memset ELL Matrix");
        for(int leveli=0; leveli<agglomeration_level; leveli++)
        {
            std::cout << "malloc and memset leveli: " << leveli << std::endl;
            // Set matrix data
            if(leveli==0)
            {

                // --- addInternalCoeffs : diag ---
                // input : d_internal_coeffs
                addInternalCoeffs(dataBase_.stream, dataBase_.num_patches, dataBase_.patch_size, 
                    d_internal_coeffs, dataBase_.d_boundary_face_cell, d_diag, dataBase_.patch_type_extropolated.data());
                
                checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_lower, d_lower, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream));
                checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_upper, d_upper, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream));
                checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_diag, d_diag, GAMGdata[leveli].nCell * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream));
                // checkCudaErrors(hipMemset(GAMGdata[leveli].d_ell_cols, 0, GAMGdata[leveli].ell_row_maxcount * GAMGdata[leveli].nCell * sizeof(int)));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_ell_values, 0, GAMGdata[leveli].ell_row_maxcount * GAMGdata[leveli].nCell * sizeof(double)));
            }
            else
            {
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_lower, 0, GAMGdata[leveli].nFace * sizeof(double)));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_upper, 0, GAMGdata[leveli].nFace * sizeof(double)));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_diag, 0, GAMGdata[leveli].nCell * sizeof(double)));
                // checkCudaErrors(hipMemset(GAMGdata[leveli].d_ell_cols, 0, GAMGdata[leveli].ell_row_maxcount * GAMGdata[leveli].nCell * sizeof(int)));
                checkCudaErrors(hipMemset(GAMGdata[leveli].d_ell_values, 0, GAMGdata[leveli].ell_row_maxcount * GAMGdata[leveli].nCell * sizeof(double)));

            }


#ifdef PARALLEL_
            if (leveli==0)
            {
                getInterfacesCoeffs(dataBase_.stream, dataBase_.num_patches, dataBase_.patch_size, 
                                    dataBase_.interfaceFlag, dataBase_.patch_type_extropolated.data(), 
                                    d_boundary_coeffs, d_internal_coeffs, 
                                    GAMGdata[0].d_interfaceBouCoeffs, GAMGdata[0].d_interfaceIntCoeffs);

                bool loadCPUCoeffdata4test = false;
                if (loadCPUCoeffdata4test){
                    int mpirank = dataBase_.localRank;
                    for(int patchi=0; patchi<GAMGdata[leveli].nPatchFaces.size(); patchi++)
                    {
                        if (GAMGdata[leveli].nPatchFaces[patchi] > 0)
                        {
                            std::stringstream filenameBouCoeff; 
                            std::stringstream filenameIntCoeff; 

                            filenameBouCoeff << "/root/0427/deepflame-dev/examples/dfLowMachFoam/cvodeIntegrator_64/h_bouCoeff_0_" << patchi << "_" << mpirank << "_Ref.txt";
                            filenameIntCoeff << "/root/0427/deepflame-dev/examples/dfLowMachFoam/cvodeIntegrator_64/h_intCoeff_0_" << patchi << "_" << mpirank << "_Ref.txt";

                            std::string filename_bouCoeff = filenameBouCoeff.str();
                            std::string filename_intCoeff = filenameIntCoeff.str();

                            std::vector<double> data_bouCoeff, data_intCoeff;

                            int offset = 0;
                            std::ifstream file_bouCoeff(filename_bouCoeff);
                            if (!file_bouCoeff.is_open()) {
                                    std::cerr << "error open filename_bouCoeff : " << filename_bouCoeff << std::endl;
                            }
                            double number_bouCoeff;
                            while (file_bouCoeff >> std::setprecision(10) >> number_bouCoeff){
                                if(offset < GAMGdata[leveli].nPatchFaces[patchi]){
                                    data_bouCoeff.push_back(number_bouCoeff);
                                }
                                offset++;
                            }
                            file_bouCoeff.close();

                            offset = 0;
                            std::ifstream file_intCoeff(filename_intCoeff);
                            if (!file_intCoeff.is_open()) {
                                    std::cerr << "error open filename_intCoeff : " << filename_intCoeff << std::endl;
                            }
                            double number_intCoeff;
                            while (file_intCoeff >> std::setprecision(10) >> number_intCoeff){
                                if(offset < GAMGdata[leveli].nPatchFaces[patchi]){
                                    data_intCoeff.push_back(number_intCoeff);
                                }
                                offset++;
                            }
                            file_intCoeff.close();

                            hipMemcpy(GAMGdata[0].d_interfaceBouCoeffs[patchi], &data_bouCoeff[0], sizeof(double)*GAMGdata[leveli].nPatchFaces[patchi], hipMemcpyHostToDevice);
                            hipMemcpy(GAMGdata[0].d_interfaceIntCoeffs[patchi], &data_intCoeff[0], sizeof(double)*GAMGdata[leveli].nPatchFaces[patchi], hipMemcpyHostToDevice);
                        }
                    }
                }
            }
            else
            {
                // Set interface coef data
                for(int patchi=0; patchi<GAMGdata[leveli].nPatchFaces.size(); patchi++)
                {
                    if (GAMGdata[leveli].nPatchFaces[patchi] > 0)
                    {
                        std::cout << "memset 0 in patch: " << patchi << std::endl;

                        checkCudaErrors(hipMemset(GAMGdata[leveli].d_interfaceBouCoeffs[patchi], 0, GAMGdata[leveli].nPatchFaces[patchi]*sizeof(double)));
                        checkCudaErrors(hipMemset(GAMGdata[leveli].d_interfaceIntCoeffs[patchi], 0, GAMGdata[leveli].nPatchFaces[patchi]*sizeof(double)));
                    
                    }
                }
            }
#endif
        }
        nvtxRangePop();

        std::cout << "=============================================" << std::endl;
        std::cout << "====== call pELLSolver->initGAMGMatrix ======" << std::endl;
        nvtxRangePushA("initGAMGMatrix()");
        pELLSolver->initGAMGMatrix(dataBase_, GAMGdata, agglomeration_level);
        nvtxRangePop();
        std::cout << "====== end pELLSolver->initGAMGMatrix =======" << std::endl;
        std::cout << "=============================================" << std::endl;

        // coarse level ldu to ell
        nvtxRangePushA("ldu to ell");
        for(int leveli=0; leveli<agglomeration_level; leveli++)
        {

            // ldu2ell
            // GAMGdata[leveli].ell_row_maxcount = dataBase_.h_ell_row_maxcount[leveli];
            // checkCudaErrors(hipMemcpy(GAMGdata[leveli].d_ell_cols, dataBase_.h_ellCols[leveli], GAMGdata[leveli].nCell * GAMGdata[leveli].ell_row_maxcount * sizeof(int), hipMemcpyHostToDevice));
            // checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_lowerAddr, &GAMGdata[leveli].lowerAddr[0], GAMGdata[leveli].nFace * sizeof(int), hipMemcpyHostToDevice, dataBase_.stream));
            // checkCudaErrors(hipMemcpyAsync(GAMGdata[leveli].d_upperAddr, &GAMGdata[leveli].upperAddr[0], GAMGdata[leveli].nFace * sizeof(int), hipMemcpyHostToDevice, dataBase_.stream));
            // checkCudaErrors(hipMemcpy(GAMGlduPtr[leveli], GAMGdata[leveli].d_lower, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice));
            // checkCudaErrors(hipMemcpy(GAMGlduPtr[leveli] + GAMGdata[leveli].nFace, GAMGdata[leveli].d_upper, GAMGdata[leveli].nFace * sizeof(double), hipMemcpyDeviceToDevice));
            nvtxRangePushA("peqn_ldu_to_ell_no_diag()");
            // peqn_ldu_to_ell_no_diag(GAMGdata[leveli].nFace * 2, GAMGldu2ellPtr[leveli], GAMGlduPtr[leveli], GAMGdata[leveli].d_ell_values);
            peqn_ldu_to_ell_no_diag(GAMGdata[leveli].nFace * 2, GAMGldu2ellPtr[leveli], GAMGdata[leveli].d_lower, GAMGdata[leveli].d_upper, GAMGdata[leveli].d_ell_values);
            nvtxRangePop();
        }
        nvtxRangePop();

#endif

#ifdef USE_GRAPH
        checkCudaErrors(hipStreamEndCapture(dataBase_.stream, &graph_pre));
        checkCudaErrors(hipGraphInstantiate(&graph_instance_pre, graph_pre, NULL, NULL, 0));
        pre_graph_created = true;
    }
    DEBUG_TRACE;
    checkCudaErrors(hipGraphLaunch(graph_instance_pre, dataBase_.stream));
#endif
    TICK_END_EVENT(pEqn assembly);

    TICK_START_EVENT;
    solve(GAMGdata, agglomeration_level);
    TICK_END_EVENT(pEqn solve);

#ifdef USE_GRAPH
    if(!post_graph_created) {
        checkCudaErrors(hipStreamBeginCapture(dataBase_.stream, hipStreamCaptureModeGlobal));
#endif
    
        TICK_START_EVENT;
        correct_boundary_conditions_scalar(dataBase_.stream, dataBase_.nccl_comm, dataBase_.neighbProcNo.data(),
                dataBase_.num_boundary_surfaces, dataBase_.num_patches, dataBase_.patch_size.data(), 
                patch_type_p.data(), dataBase_.d_boundary_delta_coeffs,
                dataBase_.d_boundary_face_cell, dataBase_.d_p, dataBase_.d_boundary_p,
                dataBase_.cyclicNeighbor.data(), dataBase_.patchSizeOffset.data(), dataBase_.d_boundary_weight);
        // update phi
        fvMtx_flux(dataBase_.stream, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces, dataBase_.d_owner, dataBase_.d_neighbor, 
                d_lower, d_upper, dataBase_.d_p, d_flux,
                dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_p.data(), 
                dataBase_.d_boundary_face_cell, d_internal_coeffs, d_boundary_coeffs, dataBase_.cyclicNeighbor.data(), 
                dataBase_.patchSizeOffset.data(), dataBase_.d_boundary_p, d_boundary_flux);
        field_add_scalar(dataBase_.stream, dataBase_.num_surfaces, d_phiHbyA, d_flux, dataBase_.d_phi, 
                dataBase_.num_boundary_surfaces, d_boundary_phiHbyA, d_boundary_flux, dataBase_.d_boundary_phi);
        // correct U
        checkCudaErrors(hipMemsetAsync(dataBase_.d_u, 0., dataBase_.cell_value_vec_bytes, dataBase_.stream));
        // TODO: may do not need to calculate boundary fields
        fvc_grad_cell_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces,
                dataBase_.d_owner, dataBase_.d_neighbor, 
                dataBase_.d_weight, dataBase_.d_sf, dataBase_.d_p, dataBase_.d_u, 
                dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_p.data(), dataBase_.d_boundary_weight,
                dataBase_.d_boundary_face_cell, dataBase_.d_boundary_p, dataBase_.d_boundary_sf, dataBase_.d_volume, true);
        scalar_field_multiply_vector_field(dataBase_.stream, dataBase_.num_cells, dataBase_.d_rAU, dataBase_.d_u, dataBase_.d_u);
        field_add_vector(dataBase_.stream, dataBase_.num_cells, dataBase_.d_HbyA, dataBase_.d_u, dataBase_.d_u, -1.);
        correct_boundary_conditions_vector(dataBase_.stream, dataBase_.nccl_comm, dataBase_.neighbProcNo.data(), dataBase_.num_boundary_surfaces, 
                dataBase_.num_cells, dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_U.data(), dataBase_.d_boundary_weight,
                dataBase_.d_boundary_face_cell, dataBase_.d_u, dataBase_.d_boundary_u, 
                dataBase_.cyclicNeighbor.data(), dataBase_.patchSizeOffset.data());
        vector_half_mag_square(dataBase_.stream, dataBase_.num_cells, dataBase_.d_u, dataBase_.d_k, dataBase_.num_boundary_surfaces, 
                dataBase_.d_boundary_u, dataBase_.d_boundary_k);
        // calculate dpdt
        fvc_ddt_scalar_field(dataBase_.stream, dataBase_.num_cells, dataBase_.rdelta_t, dataBase_.d_p, dataBase_.d_p_old, dataBase_.d_volume, dataBase_.d_dpdt, 1.);

#ifdef STREAM_ALLOCATOR
        // intermediate fields
        checkCudaErrors(hipFreeAsync(d_rhorAUf, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_phiHbyA, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_flux, dataBase_.stream));

        // boundary coeffs
        checkCudaErrors(hipFreeAsync(d_value_internal_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_value_boundary_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_gradient_internal_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_gradient_boundary_coeffs, dataBase_.stream));
        // intermediate boundary fields
        checkCudaErrors(hipFreeAsync(d_boundary_rhorAUf, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_boundary_phiHbyA, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_boundary_flux, dataBase_.stream));

        // ldu and csr
        checkCudaErrors(hipFreeAsync(d_source, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_internal_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_boundary_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_A, dataBase_.stream));
#endif
        TICK_END_EVENT(pEqn post process all);

#ifdef USE_GRAPH
        checkCudaErrors(hipStreamEndCapture(dataBase_.stream, &graph_post));
        checkCudaErrors(hipGraphInstantiate(&graph_instance_post, graph_post, NULL, NULL, 0));
        post_graph_created = true;
    }
    checkCudaErrors(hipGraphLaunch(graph_instance_post, dataBase_.stream));
#endif
    sync();
}
void dfpEqn::postProcess() {}

//void dfpEqn::getFlux()
//{
//    fvMtx_flux(dataBase_.stream, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces, dataBase_.d_owner, dataBase_.d_neighbor, 
//            d_lower, d_upper, dataBase_.d_p, d_flux,
//            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_p.data(), 
//            dataBase_.d_boundary_face_cell, d_internal_coeffs, d_boundary_coeffs, dataBase_.d_boundary_p, d_boundary_flux);
//    sync();
//}

void dfpEqn::getrhorAUf(hipStream_t stream, int num_cells, int num_surfaces,
        const int *lowerAddr, const int *upperAddr, 
        const double *weight, const double *vf1, const double *vf2, double *output, // end for internal
        int num_patches, const int *patch_size, const int *patch_type, const double *boundary_weight,
        const double *boundary_vf1, const double *boundary_vf2, double *boundary_output, double sign) 
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    fvc_interpolate_internal_multi_scalar_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_surfaces,
            lowerAddr, upperAddr, vf1, vf2, weight, output, sign);
    
    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        if (patch_size[i] == 0) continue;
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        // TODO: maybe do not need loop boundarys
        if (patch_type[i] == boundaryConditions::zeroGradient
                || patch_type[i] == boundaryConditions::fixedValue
                || patch_type[i] == boundaryConditions::calculated
                || patch_type[i] == boundaryConditions::cyclic) {
            fvc_interpolate_boundary_multi_scalar_kernel_unCouple<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset,
                    boundary_vf1, boundary_vf2, boundary_output, sign);
        } else if (patch_type[i] == boundaryConditions::processor
                    || patch_type[i] == boundaryConditions::processorCyclic) {
            fvc_interpolate_boundary_multi_scalar_kernel_processor<<<blocks_per_grid, threads_per_block, 0, stream>>>(patch_size[i], offset,
                    boundary_weight, boundary_vf1, boundary_vf2, boundary_output, sign);
            offset += 2 * patch_size[i];
            continue;
        } else {
            fprintf(stderr, "%s %d, boundaryConditions other than zeroGradient are not support yet!\n", __FILE__, __LINE__);
        }
        offset += patch_size[i];
    }
};

void dfpEqn::getphiHbyA(hipStream_t stream, int num_cells, int num_surfaces, int num_boundary_surfaces, double rDeltaT, 
        const int *lowerAddr, const int *upperAddr, 
        const double *weight, const double *u_old, const double *rho_old, const double *phi_old, const double *rho, 
        const double *rhorAUf, const double *HbyA, const double *Sf, double *output, // end for internal
        int num_patches, const int *patch_size, const int *patch_type,
        const double *boundary_Sf, const double *boundary_velocity_old, const double *boundary_rho, 
        const double *boundary_rho_old, const double *boundary_phi_old, const double *boundary_rhorAUf, const double *boundary_HbyA,
        const double *boundary_weight, double *boundary_output, double sign)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    get_phiCorr_internal_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_surfaces, lowerAddr, upperAddr, 
            phi_old, u_old, rho_old, weight, Sf, output);
    
    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        if (patch_size[i] == 0) continue;
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        if (patch_type[i] == boundaryConditions::processor
            || patch_type[i] == boundaryConditions::processorCyclic) {
            get_phiCorr_boundary_kernel_processor<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_surfaces, patch_size[i], offset,
                    boundary_Sf, boundary_velocity_old, boundary_rho_old, boundary_phi_old, boundary_weight, boundary_output);
            offset += 2 * patch_size[i];
        } else {
            get_phiCorr_boundary_kernel_zeroGradient<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_surfaces, patch_size[i], offset,
                    boundary_Sf, boundary_velocity_old, boundary_rho_old, boundary_phi_old, boundary_output);
            offset += patch_size[i];
        }
    }

    blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    get_ddtCorr_internal_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_surfaces, output, phi_old, rDeltaT, output);

    offset = 0;
    for (int i = 0; i < num_patches; i++) {
        if (patch_size[i] == 0) continue;
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        if (patch_type[i] == boundaryConditions::processor
            || patch_type[i] == boundaryConditions::processorCyclic) {
            get_ddtCorr_boundary_nonZero_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_surfaces, patch_size[i], offset, 
                    boundary_output, boundary_phi_old, rDeltaT, boundary_output);
            offset += 2 * patch_size[i];
            continue;
        }
        offset += patch_size[i];
    }

    field_multiply_scalar(stream, num_surfaces, output, rhorAUf, output, num_boundary_surfaces, boundary_output, boundary_rhorAUf, boundary_output);

    blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    multi_fvc_flux_fvc_intepolate_internal_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, num_surfaces, lowerAddr, upperAddr, 
            HbyA, rho, weight, Sf, output, sign);
    
    offset = 0;
    for (int i = 0; i < num_patches; i++) {
        if (patch_size[i] == 0) continue;
        threads_per_block = 256;
        blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;
        if (patch_type[i] == boundaryConditions::extrapolated
            || patch_type[i] == boundaryConditions::cyclic) {
            multi_fvc_flux_fvc_intepolate_boundary_kernel_zeroGradient<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_surfaces, patch_size[i], offset, 
                    boundary_Sf, boundary_HbyA, boundary_rho, boundary_output, sign);
        } else if (patch_type[i] == boundaryConditions::processor
                    || patch_type[i] == boundaryConditions::processorCyclic) {
            multi_fvc_flux_fvc_intepolate_boundary_kernel_processor<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_boundary_surfaces, patch_size[i], offset, 
                    boundary_Sf, boundary_HbyA, boundary_weight, boundary_rho, boundary_output, sign);
            offset += 2 * patch_size[i];
            continue;
        } else {
            fprintf(stderr, "%s %d, boundaryConditions other than zeroGradient are not support yet!\n", __FILE__, __LINE__);
        }
        offset += patch_size[i];
    }
}

void dfpEqn::correctionDiagMtxMultiTPsi(hipStream_t stream, int num_cells, const double *psi, const double *thermo_psi, double *diag, double *source)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_cells + threads_per_block - 1) / threads_per_block;
    correct_diag_mtx_multi_tpsi_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(num_cells, psi, thermo_psi, source, diag);
}

void dfpEqn::sync()
{
    checkCudaErrors(hipStreamSynchronize(dataBase_.stream));
}

void dfpEqn::solve(GAMGStruct *GAMGdata, int agglomeration_level)
{
#ifdef AMGX_
    double amgx_starttime , amgx_endtime;
    amgx_starttime = MPI_Wtime();
    nvtxRangePushA("AMGX::solve()");
    dataBase_.solve(num_iteration, AMGXSetting::p_setting, d_A, dataBase_.d_p, d_source);
    nvtxRangePop();
    amgx_endtime = MPI_Wtime();
    double amgx_totaltime = amgx_endtime - amgx_starttime;
    printf("amgx_totaltime = %.5f\n", amgx_totaltime);
#endif
#ifdef CSR_
    bool useGAMG = true;
    std::cout << "*** call in dfpEqn::solve() for CSR " << std::endl;
    double* d_diag_tmp;
    hipMallocAsync((void**)&d_diag_tmp, dataBase_.num_cells * sizeof(double), dataBase_.stream);
    hipMemcpyAsync(d_diag_tmp, d_diag, dataBase_.num_cells * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream);
    // hipMemcpy(dataBase_.d_csr_row_index_no_diag, GAMGdata[0].d_csr_row_index_no_diag, sizeof(int) * (GAMGdata[0].nCell+1), hipMemcpyDeviceToDevice);
    // hipMemcpy(dataBase_.d_csr_col_index_no_diag, GAMGdata[0].d_csr_col_index_no_diag, sizeof(int) * GAMGdata[0].nFace * 2, hipMemcpyDeviceToDevice);
    if (useGAMG)
    {
        double csr_starttime , csr_endtime;
        csr_starttime = MPI_Wtime();
        nvtxRangePushA("solve_useGAMG()");
        pCSRSolver -> solve_useGAMG(dataBase_, d_internal_coeffs, d_boundary_coeffs, dataBase_.patch_type_extropolated.data(), 
                        d_diag, GAMGdata[0].d_off_diag_value, d_source, dataBase_.d_p,
                        GAMGdata, agglomeration_level);
        nvtxRangePop();
        csr_endtime = MPI_Wtime();
        double csr_totaltime = csr_endtime - csr_starttime;
        printf("csr_totaltime = %.5f\n", csr_totaltime);
    }
    else
    {
        pCSRSolver -> solve(dataBase_, d_internal_coeffs, d_boundary_coeffs, dataBase_.patch_type_extropolated.data(), 
                        d_diag, GAMGdata[0].d_off_diag_value, d_source, dataBase_.d_p);
    }
    printf("peqn PCG solve end\n");
#endif
#ifdef ELL_
    bool useGAMG = true;
    std::cout << "*** call in dfpEqn::solve() for ELL " << std::endl;
    double* d_diag_tmp;
    hipMallocAsync((void**)&d_diag_tmp, dataBase_.num_cells * sizeof(double), dataBase_.stream);
    hipMemcpyAsync(d_diag_tmp, d_diag, dataBase_.num_cells * sizeof(double), hipMemcpyDeviceToDevice, dataBase_.stream);
    if (useGAMG)
    {
        double ell_starttime , ell_endtime;
        ell_starttime = MPI_Wtime();
        nvtxRangePushA("solve_useGAMG()");
        pELLSolver -> solve_useGAMG(dataBase_, d_internal_coeffs, d_boundary_coeffs, dataBase_.patch_type_extropolated.data(), 
                        d_diag, GAMGdata[0].d_ell_values, GAMGdata[0].d_ell_cols, GAMGdata[0].ell_row_maxcount, d_source, dataBase_.d_p,
                        GAMGdata, agglomeration_level);
        nvtxRangePop();
        ell_endtime = MPI_Wtime();
        double ell_totaltime = ell_endtime - ell_starttime;
        printf("ell_totaltime = %.5f\n", ell_totaltime);
    }
    else
    {
        pELLSolver -> solve(dataBase_, d_internal_coeffs, d_boundary_coeffs, dataBase_.patch_type_extropolated.data(), 
                        d_diag, GAMGdata[0].d_ell_values, GAMGdata[0].d_ell_cols, GAMGdata[0].ell_row_maxcount, d_source, dataBase_.d_p);
    }
    printf("peqn PCG solve end\n");
#endif
    num_iteration++;
}

// debug
void dfpEqn::comparerhorAUf(const double *rhorAUf, const double *boundary_rhorAUf, bool printFlag)
{
    double *h_rhorAUf = new double[dataBase_.num_surfaces];
    double *h_boundary_rhorAUf = new double[dataBase_.num_boundary_surfaces];

    checkCudaErrors(hipMemcpy(h_rhorAUf, d_rhorAUf, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_boundary_rhorAUf, d_boundary_rhorAUf, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));

    fprintf(stderr, "check h_rhorAUf\n");
    checkVectorEqual(dataBase_.num_surfaces, rhorAUf, h_rhorAUf, 1e-10, printFlag);
    fprintf(stderr, "check h_boundary_rhorAUf\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_rhorAUf, h_boundary_rhorAUf, 1e-10, printFlag);
}

void dfpEqn::comparephiHbyA(const double *phiHbyA, const double *boundary_phiHbyA, bool printFlag)
{
    double *h_phiHbyA = new double[dataBase_.num_surfaces];
    double *h_boundary_phiHbyA = new double[dataBase_.num_boundary_surfaces];

    checkCudaErrors(hipMemcpy(h_phiHbyA, d_phiHbyA, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_boundary_phiHbyA, d_boundary_phiHbyA, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));

    fprintf(stderr, "check h_phiHbyA\n");
    checkVectorEqual(dataBase_.num_surfaces, phiHbyA, h_phiHbyA, 1e-10, printFlag);
    fprintf(stderr, "check h_boundary_phiHbyA\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_phiHbyA, h_boundary_phiHbyA, 1e-10, printFlag);
}

void dfpEqn::comparephi(const double *phi, const double *boundary_phi, bool printFlag)
{
    double *h_phi = new double[dataBase_.num_surfaces];
    double *h_boundary_phi = new double[dataBase_.num_boundary_surfaces];

    checkCudaErrors(hipMemcpy(h_phi, dataBase_.d_phi, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_boundary_phi, dataBase_.d_boundary_phi, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));

    fprintf(stderr, "check h_phi\n");
    checkVectorEqual(dataBase_.num_surfaces, phi, h_phi, 1e-10, printFlag);
    fprintf(stderr, "check h_boundary_phi\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_phi, h_boundary_phi, 1e-10, printFlag);
}

void dfpEqn::comparephiFlux(const double *flux, const double *boundary_flux, bool printFlag)
{
    double *h_flux = new double[dataBase_.num_surfaces];
    double *h_boundary_flux = new double[dataBase_.num_boundary_surfaces];

    checkCudaErrors(hipMemcpy(h_flux, d_flux, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_boundary_flux, d_boundary_flux, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));

    fprintf(stderr, "check h_flux\n");
    checkVectorEqual(dataBase_.num_surfaces, flux, h_flux, 1e-10, printFlag);
    fprintf(stderr, "check h_boundary_flux\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_flux, h_boundary_flux, 1e-10, printFlag);
}

void dfpEqn::comparep(const double *p, const double *boundary_p, bool printFlag)
{
    double *h_p = new double[dataBase_.num_cells];
    double *h_boundary_p = new double[dataBase_.num_boundary_surfaces];

    checkCudaErrors(hipMemcpy(h_p, dataBase_.d_p, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_boundary_p, dataBase_.d_boundary_p, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));

    fprintf(stderr, "check h_p\n");
    checkVectorEqual(dataBase_.num_cells, p, h_p, 1e-10, printFlag);
    fprintf(stderr, "check h_boundary_p\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_p, h_boundary_p, 1e-10, printFlag);
}

void dfpEqn::compareU(const double *U, const double *boundary_U, bool printFlag)
{
    double *h_u = new double[dataBase_.num_cells * 3];
    double *h_u_ref = new double[dataBase_.num_cells * 3];
    double *h_boundary_u = new double[dataBase_.num_boundary_surfaces * 3];
    double *h_boundary_u_ref = new double[dataBase_.num_boundary_surfaces * 3];

    // permute
    for (int i = 0; i < dataBase_.num_cells; i++)
    {
        h_u_ref[dataBase_.num_cells * 0 + i] = U[i * 3 + 0];
        h_u_ref[dataBase_.num_cells * 1 + i] = U[i * 3 + 1];
        h_u_ref[dataBase_.num_cells * 2 + i] = U[i * 3 + 2];
    }
    for (int i = 0; i < dataBase_.num_boundary_surfaces; i++)
    {
        h_boundary_u_ref[dataBase_.num_boundary_surfaces * 0 + i] = boundary_U[i * 3 + 0];
        h_boundary_u_ref[dataBase_.num_boundary_surfaces * 1 + i] = boundary_U[i * 3 + 1];
        h_boundary_u_ref[dataBase_.num_boundary_surfaces * 2 + i] = boundary_U[i * 3 + 2];
    }
    checkCudaErrors(hipMemcpy(h_u, dataBase_.d_u, dataBase_.cell_value_vec_bytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_boundary_u, dataBase_.d_boundary_u, dataBase_.boundary_surface_value_vec_bytes, hipMemcpyDeviceToHost));

    // check result
    fprintf(stderr, "check h_u\n");
    checkVectorEqual(dataBase_.num_cells * 3, h_u_ref, h_u, 1e-10, printFlag);
    fprintf(stderr, "check h_boundary_u\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces * 3, h_boundary_u_ref, h_boundary_u, 1e-10, printFlag);
}

void dfpEqn::comparedpdt(const double *dpdt, bool printFlag)
{
    double *h_dpdt = new double[dataBase_.num_cells];
    checkCudaErrors(hipMemcpy(h_dpdt, dataBase_.d_dpdt, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_dpdt\n");
    checkVectorEqual(dataBase_.num_cells, dpdt, h_dpdt, 1e-10, printFlag);
}

void dfpEqn::compareResult(const double *lower, const double *upper, const double *diag, const double *source, const double *internal_coeffs, const double *boundary_coeffs,  
        bool printFlag)
{
    DEBUG_TRACE;
    std::vector<double> h_lower;
    h_lower.resize(dataBase_.num_surfaces);
    checkCudaErrors(hipMemcpy(h_lower.data(), d_lower, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_lower\n");
    checkVectorEqual(dataBase_.num_surfaces, lower, h_lower.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_upper;
    h_upper.resize(dataBase_.num_surfaces);
    checkCudaErrors(hipMemcpy(h_upper.data(), d_upper, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_upper\n");
    checkVectorEqual(dataBase_.num_surfaces, upper, h_upper.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_diag;
    h_diag.resize(dataBase_.num_cells);
    checkCudaErrors(hipMemcpy(h_diag.data(), d_diag, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_diag\n");
    checkVectorEqual(dataBase_.num_cells, diag, h_diag.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_source;
    h_source.resize(dataBase_.num_cells);
    checkCudaErrors(hipMemcpy(h_source.data(), d_source, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_source\n");
    checkVectorEqual(dataBase_.num_cells, source, h_source.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_internal_coeffs;
    h_internal_coeffs.resize(dataBase_.num_boundary_surfaces);
    checkCudaErrors(hipMemcpy(h_internal_coeffs.data(), d_internal_coeffs, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_internal_coeffs\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, internal_coeffs, h_internal_coeffs.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_boundary_coeffs;
    h_boundary_coeffs.resize(dataBase_.num_boundary_surfaces);
    checkCudaErrors(hipMemcpy(h_boundary_coeffs.data(), d_boundary_coeffs, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_boundary_coeffs\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_coeffs, h_boundary_coeffs.data(), 1e-14, printFlag);
    DEBUG_TRACE;
}

// cpu version : ldu2csr
// void dfpEqn::peqn_ldu_to_csr_no_diag
// (int row_, int num_surfaces, int *off_diag_rowptr_, double *lower, double *upper, int *lowerAddr, int *upperAddr, double *off_diag_value_)
// {
//     std::vector<int> off_diag_current_index(row_ + 1);
//     off_diag_current_index[0] = 0;
//     for(int i = 0; i < row_; ++i){
//         off_diag_current_index[i + 1] = off_diag_rowptr_[i + 1];
//     }
//     // fill non-zero value
//     for(int i = 0; i < num_surfaces; ++i){
//         int row = upperAddr[i];
//         if(row > row_)printf("overflow\n");
//         int index = off_diag_current_index[row];
//         if(index > num_surfaces * 2)printf("index overflow! index = %d\n",index);
//         off_diag_value_[index] = lower[i]; 
//         off_diag_current_index[row] += 1;
//     }
//     for(int i = 0; i < num_surfaces; ++i){
//         int row = lowerAddr[i];
//         int index = off_diag_current_index[row];
//         off_diag_value_[index] = upper[i];
//         off_diag_current_index[row] += 1;
//     }
// }

__global__ void compute_row_count(hipStream_t stream, int num_surfaces, int* d_off_diag_count, int* d_lower_count, int* d_upper_count, int* d_lowerAddr, int* d_upperAddr)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= num_surfaces)
        return;

    atomicAdd(&d_off_diag_count[d_upperAddr[index]], 1);
    atomicAdd(&d_off_diag_count[d_lowerAddr[index]], 1);

    atomicAdd(&d_lower_count[d_upperAddr[index]], 1);
    atomicAdd(&d_upper_count[d_lowerAddr[index]], 1);

}

__global__ void compute_csr_row_index
(
    hipStream_t stream, int row_, int* d_off_diag_count, int* d_csr_row_index_no_diag, 
    int* d_lower_count, int* d_upper_count, int* d_csr_lower_index, int* d_csr_upper_index
)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= row_)
        return;

    for(int i = 0; i <= index; i++){
        d_csr_row_index_no_diag[index + 1] += d_off_diag_count[i];
        d_csr_lower_index[index + 1] += d_lower_count[i];
        d_csr_upper_index[index + 1] += d_upper_count[i];
    }
}

__global__ void resort_ldu
(
    hipStream_t stream, int row_, int num_surfaces, int* d_csr_lower_index, int* d_csr_upper_index, int* d_upperAddr, 
    int* d_lowerAddr, double* d_lower, double* d_upper, double* d_resortColValue, int* d_resortColIndex,
    double* d_resortRowValue, int* d_resortRowIndex
)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= row_)
        return;

    // upper
    int offset = d_csr_upper_index[index];
    int count = 0;
    for(int j = 0; j< num_surfaces; j++){
        if(d_lowerAddr[j] == index){
            d_resortColValue[offset + count] = d_upper[j];
            d_resortColIndex[offset + count] = d_upperAddr[j];
            count++;
        }
    }

    // lower
    offset = d_csr_lower_index[index];
    count = 0;
    for(int j = 0; j< num_surfaces; j++){
        if(d_upperAddr[j] == index){
            d_resortRowValue[offset + count] = d_lower[j];
            d_resortRowIndex[offset + count] = d_lowerAddr[j];
            count++;
        }
    }
}

__global__ void compute_csr_col_index_and_value
(
    hipStream_t stream, int row_, int* d_lower_count, int* d_upper_count,
    int* d_csr_lower_index, int* d_csr_upper_index, double *d_lower, 
    double *d_upper, int *d_lowerAddr, int *d_upperAddr, double *d_off_diag_value, 
    int* d_csr_row_index_no_diag, int *d_csr_col_index_no_diag
)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= row_)
        return;

    int lsize = d_lower_count[i];
    int usize = d_upper_count[i];
    // lower
    for(int j = 0; j < lsize; j++){
        int lduloc = d_csr_lower_index[i] + j;
        int col = d_lowerAddr[lduloc];
        double value = d_lower[lduloc];
        d_csr_col_index_no_diag[d_csr_row_index_no_diag[i] + j] = col;
        d_off_diag_value[d_csr_row_index_no_diag[i] + j] = value;
    }
    // upper
    for(int j = lsize; j < (lsize + usize); j++){
        int lduloc = d_csr_upper_index[i] + j - lsize;
        int col = d_upperAddr[lduloc];
        double value = d_upper[lduloc];
        d_csr_col_index_no_diag[d_csr_row_index_no_diag[i] + j] = col;
        d_off_diag_value[d_csr_row_index_no_diag[i] + j] = value;
    }
}

void dfpEqn::peqn_ldu2csr(
    hipStream_t stream,
    int row_, 
    int num_surfaces, 
    double *d_lower, 
    double *d_upper, 
    int *d_lowerAddr, 
    int *d_upperAddr, 
    double *d_off_diag_value, 
    int *d_csr_row_index_no_diag, 
    int *d_csr_col_index_no_diag
){

    int* d_off_diag_count;
    int* d_lower_count;
    int* d_upper_count;
    int* d_csr_lower_index;
    int* d_csr_upper_index;
    hipMalloc(&d_off_diag_count, row_ * sizeof(int));
    hipMalloc(&d_lower_count, row_ * sizeof(int));
    hipMalloc(&d_upper_count, row_ * sizeof(int));
    hipMalloc(&d_csr_lower_index, (row_ + 1) * sizeof(int));
    hipMalloc(&d_csr_upper_index, (row_ + 1) * sizeof(int));
    hipMemset(d_off_diag_count, 0, row_ * sizeof(int));
    hipMemset(d_lower_count, 0, row_ * sizeof(int));
    hipMemset(d_upper_count, 0, row_ * sizeof(int));
    hipMemset(d_csr_lower_index, 0, (row_ + 1) * sizeof(int));
    hipMemset(d_csr_upper_index, 0, (row_ + 1) * sizeof(int));

    int* d_resortColIndex;
    double* d_resortColValue;
    hipMalloc(&d_resortColIndex, num_surfaces * sizeof(int));
    hipMalloc(&d_resortColValue, num_surfaces * sizeof(double));
    int* d_resortRowIndex;
    double* d_resortRowValue;
    hipMalloc(&d_resortRowIndex, num_surfaces * sizeof(int));
    hipMalloc(&d_resortRowValue, num_surfaces * sizeof(double));

    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;
    compute_row_count<<<blocks_per_grid, threads_per_block, 0, stream>>>
        (stream, num_surfaces, d_off_diag_count, d_lower_count, d_upper_count, d_lowerAddr, d_upperAddr);

    threads_per_block = 1024;
    blocks_per_grid = (row_ + threads_per_block - 1) / threads_per_block;
    compute_csr_row_index<<<blocks_per_grid, threads_per_block, 0, stream>>>
        (stream, row_, d_off_diag_count, d_csr_row_index_no_diag, d_lower_count, d_upper_count, d_csr_lower_index, d_csr_upper_index);

    threads_per_block = 1024;
    blocks_per_grid = (row_ + threads_per_block - 1) / threads_per_block;
    resort_ldu<<<blocks_per_grid, threads_per_block, 0, stream>>>
        (stream, row_, num_surfaces, d_csr_lower_index, d_csr_upper_index, d_upperAddr,
        d_lowerAddr, d_lower, d_upper, d_resortColValue, d_resortColIndex, d_resortRowValue, d_resortRowIndex);

    threads_per_block = 1024;
    blocks_per_grid = (row_ + threads_per_block - 1) / threads_per_block;
    compute_csr_col_index_and_value<<<blocks_per_grid, threads_per_block, 0, stream>>>
        (stream, row_, d_lower_count, d_upper_count, d_csr_lower_index, d_csr_upper_index, d_resortRowValue, 
        d_resortColValue, d_resortRowIndex, d_resortColIndex, d_off_diag_value, d_csr_row_index_no_diag, d_csr_col_index_no_diag);

    hipFree(d_off_diag_count);
    hipFree(d_lower_count);
    hipFree(d_upper_count);
    hipFree(d_csr_lower_index);
    hipFree(d_csr_upper_index);
    hipFree(d_resortColIndex);
    hipFree(d_resortColValue);
    hipFree(d_resortRowIndex);
    hipFree(d_resortRowValue);
}

__global__ void kernel_getInterfacesCoeffs
(
    const double* d_boundary_coeffs,
    const double* d_internal_coeffs,
    double* d_interfaceBouCoeffs,
    double* d_interfaceIntCoeffs,
    int interfaceiSize,
    int offset
)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= interfaceiSize)
        return;

    int interface_index = offset + index;
    d_interfaceBouCoeffs[index] = d_boundary_coeffs[interface_index];
    d_interfaceIntCoeffs[index] = d_internal_coeffs[interface_index];
}

void dfpEqn::getInterfacesCoeffs(
    hipStream_t stream, int num_patches, std::vector<int> patch_size, 
    int *interfaceFlag, int *patch_type, 
    const double *d_boundary_coeffs, const double *d_internal_coeffs, 
    double **d_interfaceBouCoeffs, double **d_interfaceIntCoeffs
){
    int offset = 0;
    for (int i = 0; i < num_patches; i++) {
        if (patch_size[i] == 0) continue;
        else if (interfaceFlag[i] == 0){
            (patch_type[i] == boundaryConditions::processor
                || patch_type[i] == boundaryConditions::processorCyclic) ?
                offset += 2 * patch_size[i] : offset += patch_size[i];
            continue;
        }

        size_t threads_per_block = 1024;
        size_t blocks_per_grid = (patch_size[i] + threads_per_block - 1) / threads_per_block;

        kernel_getInterfacesCoeffs<<<blocks_per_grid, threads_per_block, 0, stream>>>
            (d_boundary_coeffs, d_internal_coeffs, d_interfaceBouCoeffs[i], d_interfaceIntCoeffs[i],
             patch_size[i], offset);

        (patch_type[i] == boundaryConditions::processor
            || patch_type[i] == boundaryConditions::processorCyclic) ?
            offset += 2 * patch_size[i] : offset += patch_size[i];
    }

}


// __global__ void ldu2csr_p(int num_surfaces, int* index, double* value, double* result){
    
//     int i = blockDim.x * blockIdx.x + threadIdx.x;
//     if (i >= num_surfaces)
//         return;

//     result[i] = value[index[i]];
// }
// void dfpEqn::peqn_ldu_to_csr_no_diag
// (int num_surfaces, int* index, double* value, double* result)
// {
//     size_t threads_per_block = 1024;
//     size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;

//     ldu2csr_p<<<blocks_per_grid, threads_per_block, 0 ,dataBase_.stream>>>
//         (num_surfaces, index, value, result);
// }
__global__ void ldu2csr_p(int num_surfaces, int* index, double* value1, double* value2, double* result){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_surfaces)
        return;

    if(index[i] < num_surfaces/2){
        result[i] = value1[index[i]];
    }
    else{
        result[i] = value2[index[i] - num_surfaces/2];
    }

}
void dfpEqn::peqn_ldu_to_csr_no_diag
(int num_surfaces, int* index, double* value1, double* value2, double* result)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;

    ldu2csr_p<<<blocks_per_grid, threads_per_block, 0 ,dataBase_.stream>>>
        (num_surfaces, index, value1, value2, result);
}


__global__ void ldu2ell_p(int num_surfaces, int* index, double* value1, double* value2, double* result){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_surfaces)
        return;

    if(i < num_surfaces/2){
        result[index[i]] = value1[i];
    }
    else{
        result[index[i]] = value2[i - num_surfaces/2];
    }
}
void dfpEqn::peqn_ldu_to_ell_no_diag
(
    int num_surfaces,
    int *ldu2ellIndex,
    double *value1,
    double *value2,
    double *result
)
{
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (num_surfaces + threads_per_block - 1) / threads_per_block;

    ldu2ell_p<<<blocks_per_grid, threads_per_block, 0 ,dataBase_.stream>>>
        (num_surfaces, ldu2ellIndex, value1, value2, result);
}
