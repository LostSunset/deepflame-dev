#include "hip/hip_runtime.h"
#include "dfCSRSolver.H"
#include "dfSolverOpBase.H"
#include "dfMatrixDataBase.H"
#include <nvtx3/nvToolsExt.h>

// #define PARALLEL_
#define PRINT_

// kernel functions for PCG solver

void PCGCSRSolver::initialize(const int nCells, const size_t boundary_surface_value_bytes)
{
    // cudamalloc variables related to PCGSolver
    hipMalloc(&d_wA, nCells * sizeof(double));
    hipMalloc(&d_rA, nCells * sizeof(double));
    hipMalloc(&d_pA, nCells * sizeof(double));
    hipMalloc(&d_normFactors_tmp, nCells * sizeof(double));
    hipMalloc(&d_wArA_tmp, nCells * sizeof(double));
    hipMalloc(&d_wApA_tmp, nCells * sizeof(double));
    hipMalloc(&reduce_result, sizeof(double));
    // for parallel
    hipMalloc(&scalarSendBufList_, boundary_surface_value_bytes);
    hipMalloc(&scalarRecvBufList_, boundary_surface_value_bytes);
}

void PCGCSRSolver::initializeStream(const int nCells, const size_t boundary_surface_value_bytes, hipStream_t stream)
{
    // cudamalloc variables related to PCGSolver
    hipMallocAsync(&d_wA, nCells * sizeof(double), stream);
    hipMallocAsync(&d_rA, nCells * sizeof(double), stream);
    hipMallocAsync(&d_pA, nCells * sizeof(double), stream);
    hipMallocAsync(&d_normFactors_tmp, nCells * sizeof(double), stream);
    hipMallocAsync(&d_wArA_tmp, nCells * sizeof(double), stream);
    hipMallocAsync(&d_wApA_tmp, nCells * sizeof(double), stream);
    hipMallocAsync(&reduce_result, sizeof(double), stream);
    // for parallel
    hipMallocAsync(&scalarSendBufList_, boundary_surface_value_bytes, stream);
    hipMallocAsync(&scalarRecvBufList_, boundary_surface_value_bytes, stream);
}

void PCGCSRSolver::freeInit()
{
    hipFree(d_wA);
    hipFree(d_rA);
    hipFree(d_pA);
    hipFree(d_normFactors_tmp);
    hipFree(d_wArA_tmp);
    hipFree(d_wApA_tmp);
    hipFree(reduce_result);
    hipFree(scalarSendBufList_);
    hipFree(scalarRecvBufList_);
}

void PCGCSRSolver::freeInitStream(hipStream_t stream)
{
    hipFreeAsync(d_wA, stream);
    hipFreeAsync(d_rA, stream);
    hipFreeAsync(d_pA, stream);
    hipFreeAsync(d_normFactors_tmp, stream);
    hipFreeAsync(d_wArA_tmp, stream);
    hipFreeAsync(d_wApA_tmp, stream);
    hipFreeAsync(reduce_result, stream);
    hipFreeAsync(scalarSendBufList_, stream);
    hipFreeAsync(scalarRecvBufList_, stream);
}

void PCGCSRSolver::initializeGAMG(const dfMatrixDataBase &dataBase, const int nCells, const size_t boundary_surface_value_bytes,
                    GAMGStruct *GAMGdata_, int agglomeration_level)
{
    // cudamalloc variables related to PCGSolver
    hipMalloc(&d_wA, nCells * sizeof(double));
    hipMalloc(&d_rA, nCells * sizeof(double));
    hipMalloc(&d_pA, nCells * sizeof(double));
    hipMalloc(&d_normFactors_tmp, nCells * sizeof(double));
    hipMalloc(&d_wArA_tmp, nCells * sizeof(double));
    hipMalloc(&d_wApA_tmp, nCells * sizeof(double));
    hipMalloc(&reduce_result, sizeof(double));
    // for parallel
    hipMalloc(&scalarSendBufList_, boundary_surface_value_bytes);
    hipMalloc(&scalarRecvBufList_, boundary_surface_value_bytes);

    // preconditioner
    precond_ = new GAMGCSRPreconditioner();
    precond_->initialize(dataBase, GAMGdata_, agglomeration_level);
}

void PCGCSRSolver::initGAMGMatrix(const dfMatrixDataBase& dataBase, GAMGStruct *GAMGdata_, int agglomeration_level)
{
    // preconditioner
    precond_->agglomerateMatrix(dataBase, GAMGdata_, agglomeration_level);
}

void PCGCSRSolver::solve
(
    const dfMatrixDataBase& dataBase,
    const double* d_internal_coeffs,
    const double* d_boundary_coeffs,
    int* patch_type,
    double* diagPtr,
    const double* off_diag_value,
    const double *rhs, 
    double *psi
)
{
    printf("GPU-CSR-PCGStab::solve start --------------------------------------------\n");

    int nIterations = 0;
 
    const int row_ = dataBase.num_total_cells;
    const int nCells = dataBase.num_cells;

    double wArA = 0.; // TODO: = solverPerf.great_
    double wArAold = wArA;

    // these two int control reduce's scale
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (nCells + threads_per_block - 1) / threads_per_block;

#ifdef PRINT_    
    printf("threads_per_block = %d, blocks_per_grid = %d\n",threads_per_block, blocks_per_grid);
#endif

    double psi_ave = 0.;
    double normFactor = 0.;
    double initialResidual = 0.;
    double finalResidual = 0.;

    // --- reduce psi to get : psi_ave ---
    reduce(nCells, threads_per_block, blocks_per_grid, psi, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&psi_ave, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&psi_ave, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
    psi_ave = psi_ave / row_;

#ifdef PRINT_
    printf("psi_ave = %.10e\n",psi_ave);
#endif

    // --- addInternalCoeffs : diag ---
    // input : d_internal_coeffs
    addInternalCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size, 
        d_internal_coeffs, dataBase.d_boundary_face_cell, diagPtr, patch_type);
    
    // --- SpMV : wA ---
    // input : psi, diag
    SpMV4CSR(dataBase.stream, nCells, diagPtr, off_diag_value, dataBase.d_csr_row_index_no_diag, dataBase.d_csr_col_index_no_diag, psi, d_wA); 

#ifdef PARALLEL_
    // --- initMatrixInterfaces & updateMatrixInterfaces : wA ---
    // input : psi (neighbor's psi)
    updateMatrixInterfaces(
        dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        dataBase.neighbProcNo, dataBase.nccl_comm,
        dataBase.interfaceFlag, psi, d_wA, 
        scalarSendBufList_, scalarRecvBufList_,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

    // --- calculate : rA and pA ---
    // input : rhs, wA and diag
    calrAandpA4CSR(dataBase.stream, nCells, d_rA, rhs, d_wA, diagPtr, off_diag_value, dataBase.d_csr_row_index_no_diag, d_pA);

    // --- subBoundaryCoeffs : pA ---
    // input : d_boundary_coeffs
    subBoundaryCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, d_pA, patch_type);

    // --- calculate : pA and d_normFactors_tmp ---
    // input : psi_ave and wA, pA, rhs
    calpAandnormFactor(dataBase.stream, nCells, psi_ave, d_pA, d_normFactors_tmp, d_wA, rhs);
    
    // --- reduce d_normFactors_tmp to get : normFactor ---
    reduce(nCells, threads_per_block, blocks_per_grid, d_normFactors_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&normFactor, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&normFactor, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

    normFactor += small_;

#ifdef PRINT_
    printf("normFactor = %.10e\n",normFactor);
#endif
    
    // --- reduce abs(rA) to get : initialResidual ---
    reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
    hipMemcpyAsync(&initialResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&initialResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
        
    initialResidual = initialResidual / normFactor;

    finalResidual = initialResidual;

#ifdef PRINT_
    printf("first finalResidual = %.10e\n",finalResidual);
#endif

    if
    (
        minIter_ > 0
     || !checkConvergence(finalResidual, initialResidual, nIterations)
    ){

        do{
            wArAold = wArA;

            // TODO: precondition

            // --- calculate : d_wArA_tmp ---
            // input : wA, rA
            AmulBtoC(dataBase.stream, nCells, d_wA, d_rA, d_wArA_tmp);

            // --- reduce d_wArA_tmp to get : wArA ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_wArA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wArA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wArA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

#ifdef PRINT_
            printf("wArA = %.10e\n",wArA);
#endif

            if(nIterations == 0){
                hipMemcpyAsync(d_pA, d_wA, nCells * sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream);
            }
            else{
                double beta = wArA/wArAold;
                // --- calculate : d_pA ---
                // input : wA, beta, d_pA
                calpA(dataBase.stream, nCells, d_pA, d_wA, beta);
            }

            // --- SpMV : wA ---
            // input : pA, diag
            SpMV4CSR(dataBase.stream, nCells, diagPtr, off_diag_value, dataBase.d_csr_row_index_no_diag, dataBase.d_csr_col_index_no_diag, d_pA, d_wA);

#ifdef PARALLEL_
            // --- initMatrixInterfaces & updateMatrixInterfaces wA ---
            // input : pA (neighbor's pA)
            updateMatrixInterfaces(
                dataBase.stream, dataBase.num_patches, dataBase.patch_size,
                dataBase.neighbProcNo, dataBase.nccl_comm,
                dataBase.interfaceFlag, d_pA, d_wA, 
                scalarSendBufList_, scalarRecvBufList_,
                d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

            double wApA = 0.;
            // --- calculate : d_wApA_tmp ---
            // input : wA, pA
            AmulBtoC(dataBase.stream, nCells, d_wA, d_pA, d_wApA_tmp);

            // --- reduce d_wApA_tmp to get : wApA ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_wApA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wApA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wApA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

#ifdef PRINT_
            printf("wApA = %.10e\n",wApA);
#endif

            if (checkSingularity(abs(wApA)/normFactor)) break;

            double alpha = wArA/wApA;
            // --- calculate : psi and d_rA ---
            // input : alpha, d_pA and alpha, d_wA
            calpsiandrA(dataBase.stream, nCells, psi, d_pA, d_rA, d_wA, alpha);

            // --- reduce abs(rA) to get : finalResidual ---
            reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
            hipMemcpyAsync(&finalResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&finalResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif

            finalResidual = finalResidual / normFactor;

#ifdef PRINT_
            printf("final finalResidual = finalResidual / normFactor : %.10e\n",finalResidual);
#endif
            
        }while
        (
            (
            ++nIterations < maxIter_
            && !checkConvergence(finalResidual, initialResidual, nIterations)
            )
            || nIterations < minIter_
        );
    }

};


void PCGCSRSolver::solve_useGAMG
(
    const dfMatrixDataBase& dataBase,
    const double* d_internal_coeffs,
    const double* d_boundary_coeffs,
    int* patch_type,
    double* diagPtr,
    const double* off_diag_value,
    const double *rhs, 
    double *psi,
    GAMGStruct *GAMGdata_, 
    int agglomeration_level
)
{
    printf("GPU-CSR-PCG(GAMG)::solve start --------------------------------------------\n");

    int nIterations = 0;
 
    const int row_ = dataBase.num_total_cells;
    const int nCells = dataBase.num_cells;

    double wArA = 0.; // TODO: = solverPerf.great_
    double wArAold = wArA;

    // these two int control reduce's scale
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = (nCells + threads_per_block - 1) / threads_per_block;

#ifdef PRINT_    
    printf("threads_per_block = %d, blocks_per_grid = %d\n",threads_per_block, blocks_per_grid);
#endif

    double psi_ave = 0.;
    double normFactor = 0.;
    double initialResidual = 0.;
    double finalResidual = 0.;

    // --- reduce psi to get : psi_ave ---
    nvtxRangePushA("reduce()");
    reduce(nCells, threads_per_block, blocks_per_grid, psi, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&psi_ave, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&psi_ave, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
    nvtxRangePop();
    psi_ave = psi_ave / row_;

#ifdef PRINT_
    printf("psi_ave = %.10e\n",psi_ave);
#endif

    // --- addInternalCoeffs : diag ---
    // input : d_internal_coeffs
    nvtxRangePushA("addInternalCoeffs()");
    addInternalCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size, 
        d_internal_coeffs, dataBase.d_boundary_face_cell, diagPtr, patch_type);
    nvtxRangePop();
    
    // --- SpMV : wA ---
    // input : psi, diag
    nvtxRangePushA("SpMV4CSR");
    SpMV4CSR(dataBase.stream, nCells, diagPtr, off_diag_value, dataBase.d_csr_row_index_no_diag, dataBase.d_csr_col_index_no_diag, psi, d_wA); 
    nvtxRangePop();

#ifdef PARALLEL_
    // --- initMatrixInterfaces & updateMatrixInterfaces : wA ---
    // input : psi (neighbor's psi)
    updateMatrixInterfaces(
        dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        dataBase.neighbProcNo, dataBase.nccl_comm,
        dataBase.interfaceFlag, psi, d_wA, 
        scalarSendBufList_, scalarRecvBufList_,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

    // --- calculate : rA and pA ---
    // input : rhs, wA and diag
    nvtxRangePushA("calrAandpA4CSR()");
    calrAandpA4CSR(dataBase.stream, nCells, d_rA, rhs, d_wA, diagPtr, off_diag_value, dataBase.d_csr_row_index_no_diag, d_pA);
    nvtxRangePop();

    // --- subBoundaryCoeffs : pA ---
    // input : d_boundary_coeffs
    nvtxRangePushA("subBoundaryCoeffs()");
    subBoundaryCoeffs(dataBase.stream, dataBase.num_patches, dataBase.patch_size,
        d_boundary_coeffs, dataBase.d_boundary_face_cell, d_pA, patch_type);
    nvtxRangePop();

    // --- calculate : pA and d_normFactors_tmp ---
    // input : psi_ave and wA, pA, rhs
    nvtxRangePushA("calpAandnormFactor()");
    calpAandnormFactor(dataBase.stream, nCells, psi_ave, d_pA, d_normFactors_tmp, d_wA, rhs);
    nvtxRangePop();
    
    // --- reduce d_normFactors_tmp to get : normFactor ---
    nvtxRangePushA("reduce()");
    reduce(nCells, threads_per_block, blocks_per_grid, d_normFactors_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
    hipMemcpyAsync(&normFactor, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&normFactor, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
    nvtxRangePop();

    normFactor += small_;

#ifdef PRINT_
    printf("normFactor = %.10e\n",normFactor);
#endif
    
    // --- reduce abs(rA) to get : initialResidual ---
    nvtxRangePushA("reduce()");
    reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
    hipMemcpyAsync(&initialResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
    ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
    hipStreamSynchronize(dataBase.stream);
    hipMemcpyAsync(&initialResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
    nvtxRangePop();
        
    initialResidual = initialResidual / normFactor;

    finalResidual = initialResidual;

#ifdef PRINT_
    printf("first finalResidual = %.10e\n",finalResidual);
#endif

    if
    (
        minIter_ > 0
     || !checkConvergence(finalResidual, initialResidual, nIterations)
    ){

        do{

            nvtxRangePushA("precondition()");
            precond_->precondition(d_wA, d_rA, dataBase, GAMGdata_, agglomeration_level, scalarSendBufList_, scalarRecvBufList_);
            nvtxRangePop();

            wArAold = wArA;

            // --- calculate : d_wArA_tmp ---
            // input : wA, rA
            nvtxRangePushA("AmulBtoC()");
            AmulBtoC(dataBase.stream, nCells, d_wA, d_rA, d_wArA_tmp);
            nvtxRangePop();

            // --- reduce d_wArA_tmp to get : wArA ---
            nvtxRangePushA("reduce()");
            reduce(nCells, threads_per_block, blocks_per_grid, d_wArA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wArA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wArA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
            nvtxRangePop();

#ifdef PRINT_
            printf("wArA = %.10e\n",wArA);
#endif

            if(nIterations == 0){
                hipMemcpyAsync(d_pA, d_wA, nCells * sizeof(double), hipMemcpyDeviceToDevice, dataBase.stream);
            }
            else{
                double beta = wArA/wArAold;
                // --- calculate : d_pA ---
                // input : wA, beta, d_pA
                nvtxRangePushA("calpA()");
                calpA(dataBase.stream, nCells, d_pA, d_wA, beta);
                nvtxRangePop();
            }

            // --- SpMV : wA ---
            // input : pA, diag
            nvtxRangePushA("SpMV4CSR()");
            SpMV4CSR(dataBase.stream, nCells, diagPtr, off_diag_value, dataBase.d_csr_row_index_no_diag, dataBase.d_csr_col_index_no_diag, d_pA, d_wA);
            nvtxRangePop();

#ifdef PARALLEL_
            // --- initMatrixInterfaces & updateMatrixInterfaces wA ---
            // input : pA (neighbor's pA)
            updateMatrixInterfaces(
                dataBase.stream, dataBase.num_patches, dataBase.patch_size,
                dataBase.neighbProcNo, dataBase.nccl_comm,
                dataBase.interfaceFlag, d_pA, d_wA, 
                scalarSendBufList_, scalarRecvBufList_,
                d_boundary_coeffs, dataBase.d_boundary_face_cell, patch_type);
#endif

            double wApA = 0.;
            // --- calculate : d_wApA_tmp ---
            // input : wA, pA
            nvtxRangePushA("AmulBtoC()");
            AmulBtoC(dataBase.stream, nCells, d_wA, d_pA, d_wApA_tmp);
            nvtxRangePop();

            // --- reduce d_wApA_tmp to get : wApA ---
            nvtxRangePushA("reduce()");
            reduce(nCells, threads_per_block, blocks_per_grid, d_wApA_tmp, reduce_result, dataBase.stream, false);
#ifndef PARALLEL_
            hipMemcpyAsync(&wApA, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&wApA, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
            nvtxRangePop();

#ifdef PRINT_
            printf("wApA = %.10e\n",wApA);
#endif

            if (checkSingularity(abs(wApA)/normFactor)) break;

            double alpha = wArA/wApA;
            // --- calculate : psi and d_rA ---
            // input : alpha, d_pA and alpha, d_wA
            nvtxRangePushA("calpsiandrA()");
            calpsiandrA(dataBase.stream, nCells, psi, d_pA, d_rA, d_wA, alpha);
            nvtxRangePop();

            // --- reduce abs(rA) to get : finalResidual ---
            nvtxRangePushA("reduce()");
            reduce(nCells, threads_per_block, blocks_per_grid, d_rA, reduce_result, dataBase.stream, true);
#ifndef PARALLEL_
            hipMemcpyAsync(&finalResidual, &reduce_result[0] , sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#else
            ncclAllReduce(&reduce_result[0], &reduce_result[0], 1, ncclDouble, ncclSum, dataBase.nccl_comm, dataBase.stream);
            hipStreamSynchronize(dataBase.stream);
            hipMemcpyAsync(&finalResidual, &reduce_result[0], sizeof(double), hipMemcpyDeviceToHost, dataBase.stream);
#endif
            nvtxRangePop();

            finalResidual = finalResidual / normFactor;

#ifdef PRINT_
            printf("final finalResidual = finalResidual / normFactor : %.10e\n",finalResidual);
#endif
            
        }while
        (
            (
            ++nIterations < maxIter_
            && !checkConvergence(finalResidual, initialResidual, nIterations)
            )
            || nIterations < minIter_
        );
    }

};